#include <cstdio>
#include <cstdlib>
#include "ComputeCapability.h"
#include <hip/hip_runtime_api.h>

Cc::Cc(const hipDeviceProp_t deviceProp)
{
	this->cudaVersion = CUDART_VERSION;
	deviceCcMajor = deviceProp.major;
	deviceCcMinor = deviceProp.minor;
}

Cc::~Cc() = default;

int Cc::GetSmxBlock() const
{
	auto result = 0;
	if (cudaVersion >= 12000 && cudaVersion < 13000)
	{
		result = GetSmxBlockCuda12();
	}
	else if (cudaVersion >= 11000 && cudaVersion < 12000)
	{
		result = GetSmxBlockCuda11();
	}
	else if (cudaVersion >= 10000 && cudaVersion < 11000)
	{
		result = GetSmxBlockCuda10();
	}

	return result;
}


int Cc::GetSmxBlockCuda12() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 8:
		smxBlock = GetSmxBlockCc8(); // Ampere micro architecture CC 8.0, 8.6; Ada Lovelace - CC 8.9
		break;
	case 7:
		smxBlock = GetSmxBlockCc7(); // 7.0, 7.2: Volta; 7.5: Turing 
		break;
	case 6:
		smxBlock = GetSmxBlockCc6(); // Pascal
		break;
	case 5:
		smxBlock = GetSmxBlockCc5(); // Maxwell
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}


int Cc::GetSmxBlockCuda11() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 8:
		smxBlock = GetSmxBlockCc8(); // Ampere micro architecture CC 8.0, 8.6; Ada Lovelace - CC 8.9
		break;
	case 7:
		smxBlock = GetSmxBlockCc7(); // 7.0, 7.2: Volta; 7.5: Turing 
		break;
	case 6:
		smxBlock = GetSmxBlockCc6(); // Pascal
		break;
	case 5:
		smxBlock = GetSmxBlockCc5(); // Maxwell
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}



int Cc::GetSmxBlockCuda10() const
{
	auto smxBlock = 0;
	switch (deviceCcMajor)
	{
	case 8:
		smxBlock = GetSmxBlockCc8();
		break;
	case 7:
		smxBlock = GetSmxBlockCc7();
		break;
	case 6:
		smxBlock = GetSmxBlockCc6();
		break;
	case 5:
		smxBlock = GetSmxBlockCc5();
		break;
	case 3:
		smxBlock = GetSmxBlockCc3(); // Kepler
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc8() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
	case 0:
		smxBlock = 32;	// Tesla A100 | occupancy 100% = 32 blocks per SMX
		break;
	case 6:
	case 7:
		smxBlock = 16;	// GeForce RTX 3080 etc.; Quadro A6000 | occupancy 100% = 16 blocks per SMX
		break;
	case 8:
		smxBlock = 16;	// ZLuda
		break;
	case 9:
		smxBlock = 24;	// GeForce RTX 4090, RTX 4080 16GB; RTX 6000 Ada | occupancy 100% = 24 blocks per SMX
		break;
	default:
		Exit();
		break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc7() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
		case 0:				// CC 7.0 & 7.2, occupancy 100% = 32 blocks per SMX
		case 2:
			smxBlock = 32;
			break;
		case 5:				// CC 7.5, occupancy 100% = 16 blocks per SMX
			smxBlock = 16;
			break;
		default:			
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc6() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
		case 0:
		case 1:
		case 2:
			smxBlock = 32; //occupancy 100% = 32 blocks per SMX
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc5() const
{
	auto smxBlock = 0;
	switch (deviceCcMinor)
	{
	// TODO: There is something rot in Denmark...
//#if (CUDART_VERSION < 11000)
		case 0:
		case 2:
//#endif
		case 3:
			smxBlock = 32; //occupancy 100% = 32 blocks per SMX, instead as previous was 16 blocks per SMX which led to only 50%
			break;

		default:
			Exit();
			break;
	}

	return smxBlock;
}

int Cc::GetSmxBlockCc3() const
{
	auto smxBlock = 0;
	switch(deviceCcMinor)
	{
		//CC 3.0, 3.2, 3.5 & 3.7
		case 0:
		case 2:
		case 3:
		case 5:
		case 7:
			smxBlock = 16; //occupancy 100% = 16 blocks per SMX
			break;
		default:
			Exit();
			break;
	}

	return smxBlock;
}

void Cc::Exit() const
{
	fprintf(stderr, "Unsupported Compute Capability (CC) detected (%d.%d).\n", deviceCcMajor, deviceCcMinor);
	exit(1);
}
