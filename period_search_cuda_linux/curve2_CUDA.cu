#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"

__device__ void mrqcof_curve2(freq_context *CUDA_LCC, double a[],  
	      double *alpha, double beta[], int Inrel,int Lpoints)
{
   int l,jp,j,k,m, lnp1,lnp2,Lpoints1=Lpoints+1;
   double dy,sig2i,wt,ymod, coef1,coef,  wght,ltrial_chisq;
   int2 xx;

   
//precalc thread boundaries
    int tmph,tmpl;
	tmph=Lpoints/CUDA_BLOCK_DIM;
	if(Lpoints%CUDA_BLOCK_DIM) tmph++;
	tmpl=threadIdx.x*tmph;
	lnp1=(*CUDA_LCC).np1+tmpl;
	tmph=tmpl+tmph;
	if (tmph>Lpoints) tmph=Lpoints;
	tmpl++;

    int matmph,matmpl;
	matmph=CUDA_ma/CUDA_BLOCK_DIM;
	if(CUDA_ma%CUDA_BLOCK_DIM) matmph++;
	matmpl=threadIdx.x*matmph;
	matmph=matmpl+matmph;
	if (matmph>CUDA_ma) matmph=CUDA_ma;
	matmpl++;

    int latmph,latmpl;
	latmph=CUDA_lastone/CUDA_BLOCK_DIM;
	if(CUDA_lastone%CUDA_BLOCK_DIM) latmph++;
	latmpl=threadIdx.x*latmph;
	latmph=latmpl+latmph;
	if (latmph>CUDA_lastone) latmph=CUDA_lastone;
	latmpl++;

/*   if ((*CUDA_LCC).Lastcall != 1) always ==0
   {*/
    if (Inrel /*==1*/) 
	{
      for (jp = tmpl; jp <= tmph; jp++)
      {
        lnp1++;
			int ixx=jp+1*Lpoints1;
				/* Set the size scale coeff. deriv. explicitly zero for relative lcurves */
			(*CUDA_LCC).dytemp[ixx]=0;

			xx=tex1Dfetch(texsig,lnp1);
			coef = __hiloint2double(xx.y,xx.x) * Lpoints / (*CUDA_LCC).ave;

			double yytmp=(*CUDA_LCC).ytemp[jp];
			coef1= yytmp / (*CUDA_LCC).ave;
			(*CUDA_LCC).ytemp[jp] = coef * yytmp;

			ixx+=Lpoints1;
			for (l = 2; l <= CUDA_ma; l++,ixx+=Lpoints1)
				(*CUDA_LCC).dytemp[ixx] = coef * ((*CUDA_LCC).dytemp[ixx] - coef1 * (*CUDA_LCC).dave[l]);
	  }
	}
	__syncthreads();

   if (threadIdx.x==0) 
   {
   (*CUDA_LCC).np1+=Lpoints;
   }
   //
   lnp2=(*CUDA_LCC).np2;
   ltrial_chisq=(*CUDA_LCC).trial_chisq;

  
   if (CUDA_ia[1]) //not relative
	  { 
		  for (jp = 1; jp <= Lpoints; jp++)
		  {
			 ymod = (*CUDA_LCC).ytemp[jp];

			 int ixx=jp+matmpl*Lpoints1;
			 for (l = matmpl; l <= matmph; l++,ixx+=Lpoints1)
					(*CUDA_LCC).dyda[l] = (*CUDA_LCC).dytemp[ixx];
			 __syncthreads();

			 lnp2++;
			xx=tex1Dfetch(texsig,lnp2);
			sig2i = 1 / (__hiloint2double(xx.y,xx.x) * __hiloint2double(xx.y,xx.x));

			xx=tex1Dfetch(texWeight,lnp2);
			wght =  __hiloint2double(xx.y,xx.x); 

			xx=tex1Dfetch(texbrightness,lnp2);
			dy = __hiloint2double(xx.y,xx.x) - ymod;

			 j = 0;
			 //
			 double sig2iwght=sig2i * wght;
			 //
			 for (l = 1; l <= CUDA_lastone; l++)
			 {
				   j++;
				   wt = (*CUDA_LCC).dyda[l] * sig2iwght;
//				   k = 0;
//precalc thread boundaries
					tmph=l/CUDA_BLOCK_DIM;
					if(l%CUDA_BLOCK_DIM) tmph++;
					tmpl=threadIdx.x*tmph;
					tmph=tmpl+tmph;
					if (tmph>l) tmph=l;
					tmpl++;
				   for (m = tmpl; m <= tmph; m++)
				   {
	//				  k++;
					  alpha[j*(CUDA_mfit1)+m] = alpha[j*(CUDA_mfit1)+m] + wt * (*CUDA_LCC).dyda[m];
				   } /* m */
				   __syncthreads();
				   if (threadIdx.x==0)
				   {
					   beta[j] = beta[j] + dy * wt;
				   }
				   __syncthreads();
			 } /* l */ 
			 for (; l <= CUDA_lastma; l++)
			 {
				if(CUDA_ia[l]) 
				{
				   j++;
				   wt = (*CUDA_LCC).dyda[l] * sig2iwght;
//				   k = 0;

				   for (m = latmpl; m <= latmph; m++)
				   {
//					  k++;
					  alpha[j*(CUDA_mfit1)+m] = alpha[j*(CUDA_mfit1)+m] + wt * (*CUDA_LCC).dyda[m];
				   } /* m */
				   __syncthreads();
				   if (threadIdx.x==0)
				   {
					   k=CUDA_lastone;
					   m=CUDA_lastone+1;
					   for (; m <= l; m++)
					   {
						  if(CUDA_ia[m])
						  {
							 k++;
							 alpha[j*(CUDA_mfit1)+k] = alpha[j*(CUDA_mfit1)+k] + wt * (*CUDA_LCC).dyda[m];
						  }
					   } /* m */
					   beta[j] = beta[j] + dy * wt;
				   }
				   __syncthreads();
				  }
			 } /* l */ 
			 ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
			} /* jp */
	  }
	  else //relative ia[1]==0
	  {
		  for (jp = 1; jp <= Lpoints; jp++)
		  {
			 ymod = (*CUDA_LCC).ytemp[jp];

			 int ixx=jp+matmpl*Lpoints1;
			 for (l = matmpl; l <= matmph; l++,ixx+=Lpoints1)
					(*CUDA_LCC).dyda[l] = (*CUDA_LCC).dytemp[ixx];
			 __syncthreads();

			 lnp2++;

			xx=tex1Dfetch(texsig,lnp2);
			sig2i = 1 / (__hiloint2double(xx.y,xx.x) * __hiloint2double(xx.y,xx.x));

			xx=tex1Dfetch(texWeight,lnp2);
			wght =  __hiloint2double(xx.y,xx.x); 

			xx=tex1Dfetch(texbrightness,lnp2);
			dy = __hiloint2double(xx.y,xx.x) - ymod;

			 j = 0;
			 //
			 double sig2iwght=sig2i * wght;
			 //l==1
			 //
			 for (l = 2; l <= CUDA_lastone; l++)
			 {
				   j++;
				   wt = (*CUDA_LCC).dyda[l] * sig2iwght;
//				   k = 0;
//precalc thread boundaries
					tmph=l/CUDA_BLOCK_DIM;
					if(l%CUDA_BLOCK_DIM) tmph++;
					tmpl=threadIdx.x*tmph;
					tmph=tmpl+tmph;
					if (tmph>l) tmph=l;
					tmpl++;
				   //m==1
					if (tmpl==1) tmpl++;
				   //
				   for (m = tmpl; m <= tmph; m++)
				   {
//					  k++;
					  alpha[j*(CUDA_mfit1)+m-1] = alpha[j*(CUDA_mfit1)+m-1] + wt * (*CUDA_LCC).dyda[m];
				   } /* m */
				   __syncthreads();
				   if (threadIdx.x==0)
				   {
						beta[j] = beta[j] + dy * wt;
				   }
				  __syncthreads();
			 } /* l */ 
			 for (; l <= CUDA_lastma; l++)
			 {
				if(CUDA_ia[l]) 
				{
				   j++;
				   wt = (*CUDA_LCC).dyda[l] * sig2iwght;
//				   k = 0;

				   tmpl=latmpl;
					//m==1
					if (tmpl==1) tmpl++;
				   //
				   for (m = tmpl; m <= latmph; m++)
				   {
					  //k++;
					  alpha[j*(CUDA_mfit1)+m-1] = alpha[j*(CUDA_mfit1)+m-1] + wt * (*CUDA_LCC).dyda[m];
				   } /* m */
				   __syncthreads();
				   if (threadIdx.x==0)
				   {
					   k=CUDA_lastone-1;
					   m=CUDA_lastone+1;
					   for (; m <= l; m++)
					   {
						  if(CUDA_ia[m])
						  {
							 k++;
							 alpha[j*(CUDA_mfit1)+k] = alpha[j*(CUDA_mfit1)+k] + wt * (*CUDA_LCC).dyda[m];
						  }
					   } /* m */
					   beta[j] = beta[j] + dy * wt;
				  }
				  __syncthreads();
				}
			 } /* l */ 
			 ltrial_chisq = ltrial_chisq + dy * dy * sig2iwght;
			} /* jp */
	  }
/*     } always ==0 /* Lastcall != 1 */
         
   /*  if (((*CUDA_LCC).Lastcall == 1) && (CUDA_Inrel[i] == 1)) always ==0
        (*CUDA_LCC).Sclnw[i] = (*CUDA_LCC).Scale * CUDA_Lpoints[i] * CUDA_sig[np]/ave;*/
	if(threadIdx.x==0)
	{
		(*CUDA_LCC).np2=lnp2;
		(*CUDA_LCC).trial_chisq=ltrial_chisq;
	}
}


__global__ void CUDACalculateIter1_mrqcof1_curve2(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
    freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_curve2(CUDA_LCC,(*CUDA_LCC).cg,(*CUDA_LCC).alpha,(*CUDA_LCC).beta,Inrel,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_curve2(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_curve2(CUDA_LCC,(*CUDA_LCC).atry,(*CUDA_LCC).covar,(*CUDA_LCC).da,Inrel,Lpoints);
}
