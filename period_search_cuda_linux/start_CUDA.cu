#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include "mfile.h"
#include "globals.h"
#include "globals_CUDA.h"
#include "start_CUDA.h"
#include "declarations_CUDA.h"
#include "boinc_api.h"
#include "Start.cuh"
//#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
//#include <cuda_occupancy.h>
#include <>
#include <hip/hip_texture_types.h>

#ifdef __GNUC__
#include <time.h>
#endif

//global to all freq
__constant__ int /*CUDA_n,*/CUDA_Ncoef, CUDA_Nphpar, CUDA_Numfac, CUDA_Numfac1, CUDA_Dg_block;
__constant__ int CUDA_ia[MAX_N_PAR + 1];
__constant__ int CUDA_ma, CUDA_mfit, CUDA_mfit1, CUDA_lastone, CUDA_lastma, CUDA_ncoef0;
__device__ double CUDA_cg_first[MAX_N_PAR + 1];
__device__ double CUDA_beta_pole[N_POLES + 1];
__device__ double CUDA_lambda_pole[N_POLES + 1];
__device__ double CUDA_par[4];
//__device__ __constant__ double CUDA_cl, CUDA_Alamda_start, CUDA_Alamda_incr;
__device__ double CUDA_cl, CUDA_Alamda_start, CUDA_Alamda_incr;
__device__ int CUDA_n_iter_max, CUDA_n_iter_min, CUDA_ndata;
__device__ double CUDA_iter_diff_max;
__constant__ double CUDA_Nor[MAX_N_FAC + 1][3];
__constant__ double CUDA_conw_r;
__constant__ int CUDA_Lmax, CUDA_Mmax;
__device__ double CUDA_Fc[MAX_N_FAC + 1][MAX_LM + 1];
__device__ double CUDA_Fs[MAX_N_FAC + 1][MAX_LM + 1];
__device__ double CUDA_Pleg[MAX_N_FAC + 1][MAX_LM + 1][MAX_LM + 1];
__constant__ double CUDA_Darea[MAX_N_FAC + 1];
__device__ double CUDA_Dsph[MAX_N_FAC + 1][MAX_N_PAR + 1];
__device__ double* CUDA_ee/*[MAX_N_OBS+1][3]*/;
__device__ double* CUDA_ee0/*[MAX_N_OBS+1][3]*/;
__device__ double CUDA_tim[MAX_N_OBS + 1];
//__device__ double CUDA_brightness[MAX_N_OBS+1];
//__device__ double CUDA_sig[MAX_N_OBS+1];
//__device__ double *CUDA_Weight/*[MAX_N_OBS+1]*/;
__constant__ double CUDA_Phi_0;
__device__ int CUDA_End;
__device__ int CUDA_Is_Precalc;

//__device__ int icol;
//__device__ double pivinv;
//__shared__ int sh_icol[CUDA_BLOCK_DIM];
//__shared__ int sh_irow[CUDA_BLOCK_DIM];
//__shared__ double sh_big[CUDA_BLOCK_DIM];

texture<int2, 1> texWeight;
texture<int2, 1> texbrightness;
texture<int2, 1> texsig;

//global to one thread
__device__ freq_context* CUDA_CC;
__device__ freq_result* CUDA_FR;

texture<int2, 1> texArea;
texture<int2, 1> texDg;

int CUDA_grid_dim;
double* pee, * pee0, * pWeight;

int CUDAPrepare(int cudadev, double* beta_pole, double* lambda_pole, double* par, double cl, double Alamda_start, double Alamda_incr,
	double ee[][3], double ee0[][3], double* tim, double Phi_0, int checkex, int ndata)
{
	//init gpu
	hipSetDevice(cudadev);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	//determine gridDim
	hipDeviceProp_t deviceProp;
	int SMXBlock; // Maximum number of resident thread blocks per multiprocessor
	hipGetDeviceProperties(&deviceProp, cudadev);
	if (!checkex)
	{
		auto cudaVersion = CUDA_VERSION;
		auto totalGlobalMemory = deviceProp.totalGlobalMem / 1048576;
		auto sharedMemorySm = deviceProp.sharedMemPerMultiprocessor;
		auto sharedMemoryBlock = deviceProp.sharedMemPerBlock;

		fprintf(stderr, "CUDA version: %d\n", cudaVersion);
		fprintf(stderr, "CUDA Device number: %d\n", cudadev);
		fprintf(stderr, "CUDA Device: %s %lluMB \n", deviceProp.name, totalGlobalMemory);
		fprintf(stderr, "Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
		fprintf(stderr, "Shared memory per Block | per SM: %llu | %llu\n", sharedMemoryBlock, sharedMemorySm);
		fprintf(stderr, "Multiprocessors: %d\n", deviceProp.multiProcessorCount);

	}

	//int cudaBlockDim = CUDA_BLOCK_DIM;
	// NOTE: See this https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities , Table 15.
	// NOTE: Also this https://stackoverflow.com/questions/4391162/cuda-determining-threads-per-block-blocks-per-grid
	// NOTE: NB - Always set MaxUsedRegisters to 32 in order to achieve 100% SM occupancy (project's Configuration properties -> CUDA C/C++ -> Device)
	if (deviceProp.major == 7)
	{
		switch (deviceProp.minor)
		{
		case 0:
		case 2:
			SMXBlock = 32;	// CC 7.0 & 7.2, occupancy 100% = 32 blocks per SMX
			break;
		case 5:
			SMXBlock = 16;	// CC 7.5, occupancy 100% = 16 blocks per SMX
			break;
		default:
			SMXBlock = 16;	// unknown CC, occupancy unknown, 16 blocks per SMX
		}
	}
	else
	if (deviceProp.major == 6) //CC 6.0, 6.1 & 6.2
	{
			SMXBlock = 32; //occupancy 100% = 32 blocks per SMX
	}
	else
	if (deviceProp.major == 5) //CC 5.0, 5.2 & 5.3
	{
				SMXBlock = 32; //occupancy 100% = 32 blocks per SMX, instead as previous was 16 blocks per SMX which led to only 50%
	}
	else
	if (deviceProp.major == 3) //CC 3.0, 3.2, 3.5 & 3.7
	{
		SMXBlock = 16; //occupancy 100% = 16 blocks per SMX
	}
	/*else
	if (deviceProp.major==2) //CC 2.0 and 2.1
	{
		SMXBlock=8; //occupancy 67% = 8 blocks per SMX
	}
	else
	if ((deviceProp.major==1) && (deviceProp.major==3)) //CC 1.3
	{
		SMXBlock=8; //occupancy 50% = 8 blocks per SMX
		CUDA_BLOCK_DIM=64;
	}*/
				else
				{
					fprintf(stderr, "Unsupported Compute Capability (CC) detected (%d.%d). Supported Compute Capabilities are between 3.0 and 7.5.\n", deviceProp.major, deviceProp.minor);
					return 0;
				}

	CUDA_grid_dim = deviceProp.multiProcessorCount * SMXBlock;

	if (!checkex)
	{
		fprintf(stderr, "Resident blocks per multiprocessor: %d\n", SMXBlock);
		fprintf(stderr, "Grid dim: %d = %d*%d\n", CUDA_grid_dim, deviceProp.multiProcessorCount, SMXBlock);
		fprintf(stderr, "Block dim: %d\n", CUDA_BLOCK_DIM);
	}

	hipError_t res;

	//Global parameters
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_beta_pole), beta_pole, sizeof(double) * (N_POLES + 1));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lambda_pole), lambda_pole, sizeof(double) * (N_POLES + 1));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_par), par, sizeof(double) * 4);
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cl), &cl, sizeof(cl));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Alamda_start), &Alamda_start, sizeof(Alamda_start));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Alamda_incr), &Alamda_incr, sizeof(Alamda_incr));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Mmax), &m_max, sizeof(m_max));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Lmax), &l_max, sizeof(l_max));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_tim), tim, sizeof(double) * (MAX_N_OBS + 1));
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Phi_0), &Phi_0, sizeof(Phi_0));

	res = hipMalloc(&pWeight, (ndata + 3 + 1) * sizeof(double));
	res = hipMemcpy(pWeight, weight, (ndata + 3 + 1) * sizeof(double), hipMemcpyHostToDevice);
	res = hipBindTexture(0, texWeight, pWeight, (ndata + 3 + 1) * sizeof(double));

	res = hipMalloc(&pee, (ndata + 1) * 3 * sizeof(double));
	res = hipMemcpy(pee, ee, (ndata + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ee), &pee, sizeof(void*));

	res = hipMalloc(&pee0, (ndata + 1) * 3 * sizeof(double));
	res = hipMemcpy(pee0, ee0, (ndata + 1) * 3 * sizeof(double), hipMemcpyHostToDevice);
	res = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ee0), &pee0, sizeof(void*));

	if (res == hipSuccess) return 1; else return 0;
}

void CUDAUnprepare(void)
{
	hipUnbindTexture(texWeight);
	hipFree(pee);
	hipFree(pee0);
	hipFree(pWeight);
}

int CUDAPrecalc(double freq_start, double freq_end, double freq_step, double stop_condition, int n_iter_min, double* conw_r,
	int ndata, int* ia, int* ia_par, int* new_conw, double* cg_first, double* sig, int Numfac, double* brightness)
{
	//int* endPtr;
	int max_test_periods, iC, theEnd;
	double sum_dark_facet, ave_dark_facet;
	int i, n, m;
	int n_iter_max;
	double iter_diff_max;
	freq_result* res;
	void* pcc, * pfr, * pbrightness, * psig;

	// NOTE: max_test_periods dictates the CUDA_Grid_dim_precalc value which is actual Threads-per-Block
	/*	Cuda Compute profiler gives the following advice for almost every kernel launched:
		"Threads are executed in groups of 32 threads called warps. This kernel launch is configured to execute 16 threads per block.
		Consequently, some threads in a warp are masked off and those hardware resources are unused. Try changing the number of threads per block to be a multiple of 32 threads.
		Between 128 and 256 threads per block is a good initial range for experimentation. Use smaller thread blocks rather than one large thread block per multiprocessor
		if latency affects performance. This is particularly beneficial to kernels that frequently call __syncthreads().*/

	max_test_periods = 10; //10;
	sum_dark_facet = 0.0;
	ave_dark_facet = 0.0;

#ifdef _DEBUG
	int n_max = (int)((freq_start - freq_end) / freq_step) + 1;
	if (n_max < max_test_periods)
	{
		max_test_periods = n_max;
		fprintf(stderr, "n_max(%d) < max_test_periods (%d)\n", n_max, max_test_periods);
	}
	else
	{
		fprintf(stderr, "n_max(%d) > max_test_periods (%d)\n", n_max, max_test_periods);
	}

	fprintf(stderr, "freq_start (%.3f) - freq_end (%.3f) / freq_step (%.3f) = n_max (%d)\n", freq_start, freq_end, freq_step, n_max);
#endif

	for (i = 1; i <= n_ph_par; i++)
	{
		ia[n_coef + 3 + i] = ia_par[i];
	}

	n_iter_max = 0;
	iter_diff_max = -1;
	if (stop_condition > 1)
	{
		n_iter_max = (int)stop_condition;
		iter_diff_max = 0;
		n_iter_min = 0; /* to not overwrite the n_iter_max value */
	}
	if (stop_condition < 1)
	{
		n_iter_max = MAX_N_ITER; /* to avoid neverending loop */
		iter_diff_max = stop_condition;
	}

	hipError_t err;
	int isPrecalc = 1;
	/*int i_col, sh_icol_local[CUDA_BLOCK_DIM], sh_irow_local[CUDA_BLOCK_DIM];
	double piv_inv, sh_big_local[CUDA_BLOCK_DIM];*/

	//here move data to device
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Ncoef), &n_coef, sizeof(n_coef));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nphpar), &n_ph_par, sizeof(n_ph_par));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac), &Numfac, sizeof(Numfac));
	m = Numfac + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac1), &m, sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ia), ia, sizeof(int) * (MAX_N_PAR + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cg_first), cg_first, sizeof(double) * (MAX_N_PAR + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_max), &n_iter_max, sizeof(n_iter_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_min), &n_iter_min, sizeof(n_iter_min));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ndata), &ndata, sizeof(ndata));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_iter_diff_max), &iter_diff_max, sizeof(iter_diff_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_conw_r), conw_r, sizeof(conw_r));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nor), normal, sizeof(double) * (MAX_N_FAC + 1) * 3);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fc), f_c, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fs), f_s, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Pleg), pleg, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Darea), d_area, sizeof(double) * (MAX_N_FAC + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dsph), d_sphere, sizeof(double) * (MAX_N_FAC + 1) * (MAX_N_PAR + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Is_Precalc), &isPrecalc, sizeof isPrecalc, 0, hipMemcpyHostToDevice);
	/*hipMemcpyToSymbol(HIP_SYMBOL(icol), &i_col, sizeof(i_col));
	hipMemcpyToSymbol(HIP_SYMBOL(pivinv), &piv_inv, sizeof(piv_inv));
	hipMemcpyToSymbol(HIP_SYMBOL(sh_icol), sh_icol_local, sizeof(int) * CUDA_BLOCK_DIM);
	hipMemcpyToSymbol(HIP_SYMBOL(sh_irow), sh_irow_local, sizeof(int) * CUDA_BLOCK_DIM);
	hipMemcpyToSymbol(HIP_SYMBOL(sh_big), sh_big_local, sizeof(double) * CUDA_BLOCK_DIM);*/

	err = hipMalloc(&pbrightness, (ndata + 1) * sizeof(double));
	err = hipMemcpy(pbrightness, brightness, (ndata + 1) * sizeof(double), hipMemcpyHostToDevice);
	err = hipBindTexture(0, texbrightness, pbrightness, (ndata + 1) * sizeof(double));

	err = hipMalloc(&psig, (ndata + 1) * sizeof(double));
	err = hipMemcpy(psig, sig, (ndata + 1) * sizeof(double), hipMemcpyHostToDevice);
	err = hipBindTexture(0, texsig, psig, (ndata + 1) * sizeof(double));
	if (err) printf("Error: %s\n", hipGetErrorString(err));

	/* number of fitted parameters */
	int lmfit = 0, llastma = 0, llastone = 1, ma = n_coef + 5 + n_ph_par;
	for (m = 1; m <= ma; m++)
	{
		if (ia[m])
		{
			lmfit++;
			llastma = m;
		}
	}
	llastone = 1;
	for (m = 2; m <= llastma; m++) //ia[1] is skipped because ia[1]=0 is acceptable inside mrqcof
	{
		if (!ia[m]) break;
		llastone = m;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ma), &ma, sizeof(ma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit), &lmfit, sizeof(lmfit));
	m = lmfit + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit1), &m, sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastma), &llastma, sizeof(llastma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastone), &llastone, sizeof(llastone));
	m = ma - 2 - n_ph_par;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ncoef0), &m, sizeof(m));

	int CUDA_Grid_dim_precalc = CUDA_grid_dim;
	if (max_test_periods < CUDA_Grid_dim_precalc)
	{
		CUDA_Grid_dim_precalc = max_test_periods;
#ifdef _DEBUG
		fprintf(stderr, "CUDA_Grid_dim_precalc = %d\n", CUDA_Grid_dim_precalc);
#endif
	}

	err = hipMalloc(&pcc, CUDA_Grid_dim_precalc * sizeof(freq_context));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_CC), &pcc, sizeof(pcc));
	err = hipMalloc(&pfr, CUDA_Grid_dim_precalc * sizeof(freq_result));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_FR), &pfr, sizeof(pfr));

	m = (Numfac + 1) * (n_coef + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dg_block), &m, sizeof(m));

	double* pa, * pg, * pal, * pco, * pdytemp, * pytemp;

	err = hipMalloc(&pa, CUDA_Grid_dim_precalc * (Numfac + 1) * sizeof(double));
	err = hipBindTexture(0, texArea, pa, CUDA_Grid_dim_precalc * (Numfac + 1) * sizeof(double));
	err = hipMalloc(&pg, CUDA_Grid_dim_precalc * (Numfac + 1) * (n_coef + 1) * sizeof(double));
	err = hipBindTexture(0, texDg, pg, CUDA_Grid_dim_precalc * (Numfac + 1) * (n_coef + 1) * sizeof(double));
	err = hipMalloc(&pal, CUDA_Grid_dim_precalc * (lmfit + 1) * (lmfit + 1) * sizeof(double));
	err = hipMalloc(&pco, CUDA_Grid_dim_precalc * (lmfit + 1) * (lmfit + 1) * sizeof(double));
	err = hipMalloc(&pdytemp, CUDA_Grid_dim_precalc * (max_l_points + 1) * (ma + 1) * sizeof(double));
	err = hipMalloc(&pytemp, CUDA_Grid_dim_precalc * (max_l_points + 1) * sizeof(double));

	for (m = 0; m < CUDA_Grid_dim_precalc; m++)
	{
		freq_context ps;
		ps.Area = &pa[m * (Numfac + 1)];
		ps.Dg = &pg[m * (Numfac + 1) * (n_coef + 1)];
		ps.alpha = &pal[m * (lmfit + 1) * (lmfit + 1)];
		ps.covar = &pco[m * (lmfit + 1) * (lmfit + 1)];
		ps.dytemp = &pdytemp[m * (max_l_points + 1) * (ma + 1)];
		ps.ytemp = &pytemp[m * (max_l_points + 1)];
		freq_context* pt = &((freq_context*)pcc)[m];
		err = hipMemcpy(pt, &ps, sizeof(void*) * 6, hipMemcpyHostToDevice);
	}

	res = (freq_result*)malloc(CUDA_Grid_dim_precalc * sizeof(freq_result));

	for (n = 1; n <= max_test_periods; n += CUDA_Grid_dim_precalc)
	{
		CudaCalculatePrepare<<<CUDA_Grid_dim_precalc, 1>>>(n, max_test_periods, freq_start, freq_step);
		err = hipDeviceSynchronize();

		for (m = 1; m <= N_POLES; m++)
		{
			//zero global End signal
			theEnd = 0;
			hipMemcpyToSymbol(HIP_SYMBOL(CUDA_End), &theEnd, sizeof(theEnd), 0, hipMemcpyHostToDevice);
			//hipGetSymbolAddress((void**)&endPtr, CUDA_End);
			//
			CudaCalculatePreparePole<<<CUDA_Grid_dim_precalc, 1>>>(m);
			//
#ifdef _DEBUG
			printf(".");
#endif
			auto count = 0;
			while (!theEnd)
			{
				count++;
				CudaCalculateIter1Begin<<<CUDA_Grid_dim_precalc, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqcof1Start<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>();
				for (iC = 1; iC < l_curves; iC++)
				{
					CudaCalculateIter1Mrqcof1Matrix<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(l_points[iC]);
					CudaCalculateIter1Mrqcof1Curve1<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
					CudaCalculateIter1Mrqcof1Curve2<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
				}
				CudaCalculateIter1Mrqcof1Curve1Last<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof1Curve2<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof1End<<<CUDA_Grid_dim_precalc, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqmin1End<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>();
				//mrqcof
				CudaCalculateIter1Mrqcof2Start<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>();
				for (iC = 1; iC < l_curves; iC++)
				{
					CudaCalculateIter1Mrqcof2Matrix<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(l_points[iC]);
					CudaCalculateIter1Mrqcof2Curve1<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
					CudaCalculateIter1Mrqcof2Curve2<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
				}
				CudaCalculateIter1Mrqcof2Curve1Last<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof2Curve2<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof2End<<<CUDA_Grid_dim_precalc, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqmin2End<<<CUDA_Grid_dim_precalc, 1>>>();
				CudaCalculateIter2<<<CUDA_Grid_dim_precalc, CUDA_BLOCK_DIM>>>();
				//err=hipDeviceSynchronize(); memcpy is synchro itself
				err = hipDeviceSynchronize();
				//hipMemcpy(&theEnd, endPtr, sizeof(theEnd), hipMemcpyDeviceToHost);
				hipMemcpyFromSymbol(&theEnd, HIP_SYMBOL(CUDA_End), sizeof(theEnd), 0, hipMemcpyDeviceToHost);
				theEnd = theEnd == CUDA_Grid_dim_precalc;

				//break;//debug
			}
			CudaCalculateFinishPole<<<CUDA_Grid_dim_precalc, 1>>>();
			err = hipDeviceSynchronize();
			//			err=hipMemcpyFromSymbol(&res,HIP_SYMBOL(CUDA_FR),sizeof(freq_result)*CUDA_Grid_dim_precalc);
			//			err=hipMemcpyFromSymbol(&resc,HIP_SYMBOL(CUDA_CC),sizeof(freq_context)*CUDA_Grid_dim_precalc);
						//break; //debug
		}
		printf("\n");

		CudaCalculateFinish<<<CUDA_Grid_dim_precalc, 1>>>();
		//err=hipDeviceSynchronize(); memcpy is synchro itself

		//read results here
		err = hipMemcpy(res, pfr, sizeof(freq_result) * CUDA_Grid_dim_precalc, hipMemcpyDeviceToHost);

		for (m = 1; m <= CUDA_Grid_dim_precalc; m++)
		{
			if (res[m - 1].isReported == 1)
				sum_dark_facet = sum_dark_facet + res[m - 1].dark_best;
		}
	} /* period loop */

	isPrecalc = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Is_Precalc), &isPrecalc, sizeof(isPrecalc), 0, hipMemcpyHostToDevice);

	hipUnbindTexture(texArea);
	hipUnbindTexture(texDg);
	hipUnbindTexture(texbrightness);
	hipUnbindTexture(texsig);
	hipFree(pa);
	hipFree(pg);
	hipFree(pal);
	hipFree(pco);
	hipFree(pdytemp);
	hipFree(pytemp);
	hipFree(pcc);
	hipFree(pfr);
	hipFree(pbrightness);
	hipFree(psig);

	free((void*)res);

	ave_dark_facet = sum_dark_facet / max_test_periods;

	if (ave_dark_facet < 1.0)
		*new_conw = 1; /* new correct conwexity weight */
	if (ave_dark_facet >= 1.0)
		*conw_r = *conw_r * 2; /* still not good */

	return 1;
}

int CUDAStart(int n_start_from, double freq_start, double freq_end, double freq_step, double stop_condition, int n_iter_min, double conw_r,
	int ndata, int* ia, int* ia_par, double* cg_first, MFILE& mf, double escl, double* sig, int Numfac, double* brightness)
{
	int retval, i, n, m, iC, n_max = (int)((freq_start - freq_end) / freq_step) + 1;
	int n_iter_max, theEnd, LinesWritten;
	double iter_diff_max;
	freq_result* res;
	void* pcc, * pfr, * pbrightness, * psig;
	char buf[256];

	for (i = 1; i <= n_ph_par; i++)
	{
		ia[n_coef + 3 + i] = ia_par[i];
	}

	n_iter_max = 0;
	iter_diff_max = -1;
	if (stop_condition > 1)
	{
		n_iter_max = (int)stop_condition;
		iter_diff_max = 0;
		n_iter_min = 0; /* to not overwrite the n_iter_max value */
	}
	if (stop_condition < 1)
	{
		n_iter_max = MAX_N_ITER; /* to avoid neverending loop */
		iter_diff_max = stop_condition;
	}

	hipError_t err;

	/*int i_col, sh_icol_local[CUDA_BLOCK_DIM], sh_irow_local[CUDA_BLOCK_DIM];
	double piv_inv, sh_big_local[CUDA_BLOCK_DIM];*/

	//here move data to device
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Ncoef), &n_coef, sizeof(n_coef));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nphpar), &n_ph_par, sizeof(n_ph_par));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac), &Numfac, sizeof(Numfac));
	m = Numfac + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac1), &m, sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ia), ia, sizeof(int) * (MAX_N_PAR + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cg_first), cg_first, sizeof(double) * (MAX_N_PAR + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_max), &n_iter_max, sizeof(n_iter_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_min), &n_iter_min, sizeof(n_iter_min));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ndata), &ndata, sizeof(ndata));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_iter_diff_max), &iter_diff_max, sizeof(iter_diff_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_conw_r), &conw_r, sizeof(conw_r));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nor), normal, sizeof(double) * (MAX_N_FAC + 1) * 3);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fc), f_c, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fs), f_s, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Pleg), pleg, sizeof(double) * (MAX_N_FAC + 1) * (MAX_LM + 1) * (MAX_LM + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Darea), d_area, sizeof(double) * (MAX_N_FAC + 1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dsph), d_sphere, sizeof(double) * (MAX_N_FAC + 1) * (MAX_N_PAR + 1));
	/*hipMemcpyToSymbol(HIP_SYMBOL(icol), &i_col, sizeof(i_col));
	hipMemcpyToSymbol(HIP_SYMBOL(pivinv), &piv_inv, sizeof(piv_inv));
	hipMemcpyToSymbol(HIP_SYMBOL(sh_icol), sh_icol_local, sizeof(int) * CUDA_BLOCK_DIM);
	hipMemcpyToSymbol(HIP_SYMBOL(sh_irow), sh_irow_local, sizeof(int) * CUDA_BLOCK_DIM);
	hipMemcpyToSymbol(HIP_SYMBOL(sh_big), sh_big_local, sizeof(double) * CUDA_BLOCK_DIM);*/


	err = hipMalloc(&pbrightness, (ndata + 1) * sizeof(double));
	err = hipMemcpy(pbrightness, brightness, (ndata + 1) * sizeof(double), hipMemcpyHostToDevice);
	err = hipBindTexture(0, texbrightness, pbrightness, (ndata + 1) * sizeof(double));

	err = hipMalloc(&psig, (ndata + 1) * sizeof(double));
	err = hipMemcpy(psig, sig, (ndata + 1) * sizeof(double), hipMemcpyHostToDevice);
	err = hipBindTexture(0, texsig, psig, (ndata + 1) * sizeof(double));
	if (err) printf("Error: %s", hipGetErrorString(err));

	/* number of fitted parameters */
	int lmfit = 0, llastma = 0, llastone = 1, ma = n_coef + 5 + n_ph_par;
	for (m = 1; m <= ma; m++)
	{
		if (ia[m])
		{
			lmfit++;
			llastma = m;
		}
	}
	llastone = 1;
	for (m = 2; m <= llastma; m++) //ia[1] is skipped because ia[1]=0 is acceptable inside mrqcof
	{
		if (!ia[m]) break;
		llastone = m;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ma), &ma, sizeof(ma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit), &lmfit, sizeof(lmfit));
	m = lmfit + 1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit1), &m, sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastma), &llastma, sizeof(llastma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastone), &llastone, sizeof(llastone));
	m = ma - 2 - n_ph_par;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ncoef0), &m, sizeof(m));

	err = hipMalloc(&pcc, CUDA_grid_dim * sizeof(freq_context));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_CC), &pcc, sizeof(pcc));
	err = hipMalloc(&pfr, CUDA_grid_dim * sizeof(freq_result));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_FR), &pfr, sizeof(pfr));

	m = (Numfac + 1) * (n_coef + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dg_block), &m, sizeof(m));

	double* pa, * pg, * pal, * pco, * pdytemp, * pytemp;

	err = hipMalloc(&pa, CUDA_grid_dim * (Numfac + 1) * sizeof(double));
	err = hipBindTexture(0, texArea, pa, CUDA_grid_dim * (Numfac + 1) * sizeof(double));
	err = hipMalloc(&pg, CUDA_grid_dim * (Numfac + 1) * (n_coef + 1) * sizeof(double));
	err = hipBindTexture(0, texDg, pg, CUDA_grid_dim * (Numfac + 1) * (n_coef + 1) * sizeof(double));
	err = hipMalloc(&pal, CUDA_grid_dim * (lmfit + 1) * (lmfit + 1) * sizeof(double));
	err = hipMalloc(&pco, CUDA_grid_dim * (lmfit + 1) * (lmfit + 1) * sizeof(double));
	err = hipMalloc(&pdytemp, CUDA_grid_dim * (max_l_points + 1) * (ma + 1) * sizeof(double));
	err = hipMalloc(&pytemp, CUDA_grid_dim * (max_l_points + 1) * sizeof(double));

	for (m = 0; m < CUDA_grid_dim; m++)
	{
		freq_context ps;
		ps.Area = &pa[m * (Numfac + 1)];
		ps.Dg = &pg[m * (Numfac + 1) * (n_coef + 1)];
		ps.alpha = &pal[m * (lmfit + 1) * (lmfit + 1)];
		ps.covar = &pco[m * (lmfit + 1) * (lmfit + 1)];
		ps.dytemp = &pdytemp[m * (max_l_points + 1) * (ma + 1)];
		ps.ytemp = &pytemp[m * (max_l_points + 1)];
		freq_context* pt = &((freq_context*)pcc)[m];
		err = hipMemcpy(pt, &ps, sizeof(void*) * 6, hipMemcpyHostToDevice);
	}

	res = (freq_result*)malloc(CUDA_grid_dim * sizeof(freq_result));

	//int firstreport = 0;//beta debug

	for (n = n_start_from; n <= n_max; n += CUDA_grid_dim)
	{
		auto fractionDone = (double)n / (double)n_max;
		boinc_fraction_done(fractionDone);

#if _DEBUG
		float fraction = fractionDone * 100;
		std::time_t t = std::time(nullptr);   // get time now
		std::tm* now = std::localtime(&t);

		printf("%02d:%02d:%02d | Fraction done: %.3f%%\n", now->tm_hour, now->tm_min, now->tm_sec, fraction);
		fprintf(stderr, "%02d:%02d:%02d | Fraction done: %.3f%%\n", now->tm_hour, now->tm_min, now->tm_sec, fraction);
#endif

		CudaCalculatePrepare<<<CUDA_grid_dim, 1>>>(n, n_max, freq_start, freq_step);
		err = hipDeviceSynchronize();

		for (m = 1; m <= N_POLES; m++)
		{
			//zero global End signal
			theEnd = 0;
			hipMemcpyToSymbol(HIP_SYMBOL(CUDA_End), &theEnd, sizeof(theEnd));
			//
			CudaCalculatePreparePole<<<CUDA_grid_dim, 1>>>(m);
			//
			while (!theEnd)
			{
				CudaCalculateIter1Begin<<<CUDA_grid_dim, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqcof1Start<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>();
				for (iC = 1; iC < l_curves; iC++)
				{
					CudaCalculateIter1Mrqcof1Matrix<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(l_points[iC]);
					CudaCalculateIter1Mrqcof1Curve1<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
					CudaCalculateIter1Mrqcof1Curve2<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
				}
				CudaCalculateIter1Mrqcof1Curve1Last<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof1Curve2<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof1End<<<CUDA_grid_dim, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqmin1End<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>();
				//mrqcof
				CudaCalculateIter1Mrqcof2Start<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>();
				for (iC = 1; iC < l_curves; iC++)
				{
					CudaCalculateIter1Mrqcof2Matrix<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(l_points[iC]);
					CudaCalculateIter1Mrqcof2Curve1<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
					CudaCalculateIter1Mrqcof2Curve2<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[iC], l_points[iC]);
				}
				CudaCalculateIter1Mrqcof2Curve1Last<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof2Curve2<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>(in_rel[l_curves], l_points[l_curves]);
				CudaCalculateIter1Mrqcof2End<<<CUDA_grid_dim, 1>>>();
				//mrqcof
				CudaCalculateIter1Mrqmin2End<<<CUDA_grid_dim, 1>>>();
				CudaCalculateIter2<<<CUDA_grid_dim, CUDA_BLOCK_DIM>>>();
				//err=hipDeviceSynchronize(); memcpy is synchro itself
				hipMemcpyFromSymbol(&theEnd, HIP_SYMBOL(CUDA_End), sizeof(theEnd));
				theEnd = theEnd == CUDA_grid_dim;

				//break;//debug
			}
			CudaCalculateFinishPole<<<CUDA_grid_dim, 1>>>();
			err = hipDeviceSynchronize();
			//			err=hipMemcpyFromSymbol(&res,HIP_SYMBOL(CUDA_FR),sizeof(freq_result)*CUDA_grid_dim);
			//			err=hipMemcpyFromSymbol(&resc,HIP_SYMBOL(CUDA_CC),sizeof(freq_context)*CUDA_grid_dim);
						//break; //debug
		}

		CudaCalculateFinish<<<CUDA_grid_dim, 1>>>();
		//err=hipDeviceSynchronize(); memcpy is synchro itself

		//read results here
		err = hipMemcpy(res, pfr, sizeof(freq_result) * CUDA_grid_dim, hipMemcpyDeviceToHost);

		LinesWritten = 0;
		for (m = 1; m <= CUDA_grid_dim; m++)
		{
			if (res[m - 1].isReported == 1)
			{
				LinesWritten++;
				/* output file */
				if ((n == 1) && (m == 1))
					mf.printf("%.8f  %.6f  %.6f %4.1f %4.0f %4.0f\n", 24 * res[m - 1].per_best, res[m - 1].dev_best, res[m - 1].dev_best * res[m - 1].dev_best * (ndata - 3), conw_r * escl * escl, res[m - 1].la_best, res[m - 1].be_best);
				else
					mf.printf("%.8f  %.6f  %.6f %4.1f %4.0f %4.0f\n", 24 * res[m - 1].per_best, res[m - 1].dev_best, res[m - 1].dev_best * res[m - 1].dev_best * (ndata - 3), res[m - 1].dark_best, res[m - 1].la_best, res[m - 1].be_best);
			}
		}
		if (boinc_time_to_checkpoint() || boinc_is_standalone()) {
			retval = DoCheckpoint(mf, (n - 1) + LinesWritten, 1, conw_r); //zero lines
			if (retval) { fprintf(stderr, "%s APP: period_search checkpoint failed %d\n", boinc_msg_prefix(buf, sizeof(buf)), retval); exit(retval); }
			boinc_checkpoint_completed();
		}

		//		break;//debug
	} /* period loop */

	hipUnbindTexture(texArea);
	hipUnbindTexture(texDg);
	hipUnbindTexture(texbrightness);
	hipUnbindTexture(texsig);
	hipFree(pa);
	hipFree(pg);
	hipFree(pal);
	hipFree(pco);
	hipFree(pdytemp);
	hipFree(pytemp);
	hipFree(pcc);
	hipFree(pfr);
	hipFree(pbrightness);
	hipFree(psig);

	free((void*)res);

	return 1;
}
