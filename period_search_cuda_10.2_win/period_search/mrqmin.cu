#include "hip/hip_runtime.h"
/* N.B. The foll. L-M routines are modified versions of Press et al. 
   converted from Mikko's fortran code

   8.11.2006
*/

#include <hip/hip_runtime.h>
#include "globals_CUDA.h"
#include "declarations_CUDA.h"

__device__ int mrqmin_1_end(freq_context *CUDA_LCC)
{     

   int j, k, l, err_code;
//precalc thread boundaries
    int tmph,tmpl;
	tmph=CUDA_ma/CUDA_BLOCK_DIM;
	if(CUDA_ma%CUDA_BLOCK_DIM) tmph++;
	tmpl=threadIdx.x*tmph;
	tmph=tmpl+tmph;
	if (tmph>CUDA_ma) tmph=CUDA_ma;
	tmpl++;
//
    int brtmph,brtmpl;
	brtmph=CUDA_mfit/CUDA_BLOCK_DIM;
	if(CUDA_mfit%CUDA_BLOCK_DIM) brtmph++;
	brtmpl=threadIdx.x*brtmph;
	brtmph=brtmpl+brtmph;
	if (brtmph>CUDA_mfit) brtmph=CUDA_mfit;
	brtmpl++;

      if((*CUDA_LCC).isAlamda)
      {
         for (j = tmpl; j <= tmph; j++)
            (*CUDA_LCC).atry[j] = (*CUDA_LCC).cg[j];
		 __syncthreads();
      }

      for (j = brtmpl; j <= brtmph; j++)
      {
		  int ixx=j*CUDA_mfit1+1;
         for (k = 1; k <= CUDA_mfit; k++,ixx++)
            (*CUDA_LCC).covar[ixx] = (*CUDA_LCC).alpha[ixx];
         (*CUDA_LCC).covar[j*CUDA_mfit1+j] = (*CUDA_LCC).alpha[j*CUDA_mfit1+j] * (1 + (*CUDA_LCC).Alamda);
         (*CUDA_LCC).da[j] = (*CUDA_LCC).beta[j];
      }
	  __syncthreads();

		err_code = gauss_errc(CUDA_LCC,CUDA_mfit,(*CUDA_LCC).da);

//     __syncthreads(); inside gauss

	  if (threadIdx.x==0)
	  {

//		if (err_code != 0) return(err_code); bacha na sync threads

	    j = 0;
		for (l = 1; l <= CUDA_ma; l++)
        if(CUDA_ia[l]) 
		{
           j++;
           (*CUDA_LCC).atry[l] = (*CUDA_LCC).cg[l] + (*CUDA_LCC).da[j];
        }
	  }
	  __syncthreads();
	           
    return(err_code);
}

__device__ void mrqmin_2_end(freq_context *CUDA_LCC, int ia[], int ma)
{     
   int j, k, l;
   
   if ((*CUDA_LCC).Chisq < (*CUDA_LCC).Ochisq)
   {
      (*CUDA_LCC).Alamda = (*CUDA_LCC).Alamda / CUDA_Alamda_incr;
      for (j = 1; j <= CUDA_mfit; j++)
      {
         for (k = 1; k <= CUDA_mfit; k++)
            (*CUDA_LCC).alpha[j*CUDA_mfit1+k] = (*CUDA_LCC).covar[j*CUDA_mfit1+k];
         (*CUDA_LCC).beta[j] = (*CUDA_LCC).da[j];
      }
      for (l = 1; l <= ma; l++)
         (*CUDA_LCC).cg[l] = (*CUDA_LCC).atry[l];
   }
   else
   {
      (*CUDA_LCC).Alamda = CUDA_Alamda_incr * (*CUDA_LCC).Alamda;
      (*CUDA_LCC).Chisq = (*CUDA_LCC).Ochisq;
   }

    return;
}

