#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "mfile.h"
#include "globals.h"
#include "globals_CUDA.h"
#include "start_CUDA.h"
#include "declarations_CUDA.h"
#include "boinc_api.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_occupancy.h>
#include <>

#ifdef __GNUC__
#include <time.h>
#endif

//global to all freq
__constant__ int /*CUDA_n,*/CUDA_Ncoef,CUDA_Nphpar,CUDA_Numfac,CUDA_Numfac1,CUDA_Dg_block;
__constant__ int CUDA_ia[MAX_N_PAR+1];
__constant__ int CUDA_ma,CUDA_mfit,CUDA_mfit1,CUDA_lastone,CUDA_lastma,CUDA_ncoef0;
__device__ double CUDA_cg_first[MAX_N_PAR+1];
__device__ double CUDA_beta_pole[N_POLES+1];
__device__ double CUDA_lambda_pole[N_POLES+1];
__device__ double CUDA_par[4];
__device__ double CUDA_cl,CUDA_Alamda_start,CUDA_Alamda_incr;
__device__ int CUDA_n_iter_max,CUDA_n_iter_min,CUDA_ndata;
__device__ double CUDA_iter_diff_max;
__constant__ double CUDA_Nor[MAX_N_FAC+1][3];
__constant__ double CUDA_conw_r;
__constant__ int CUDA_Lmax,CUDA_Mmax;
__device__ double CUDA_Fc[MAX_N_FAC+1][MAX_LM+1];
__device__ double CUDA_Fs[MAX_N_FAC+1][MAX_LM+1];
__device__ double CUDA_Pleg[MAX_N_FAC+1][MAX_LM+1][MAX_LM+1];
__constant__ double CUDA_Darea[MAX_N_FAC+1];
__device__ double CUDA_Dsph[MAX_N_FAC+1][MAX_N_PAR+1];
__device__ double *CUDA_ee/*[MAX_N_OBS+1][3]*/;
__device__ double *CUDA_ee0/*[MAX_N_OBS+1][3]*/;
__device__ double CUDA_tim[MAX_N_OBS+1];
//__device__ double CUDA_brightness[MAX_N_OBS+1];
//__device__ double CUDA_sig[MAX_N_OBS+1];
//__device__ double *CUDA_Weight/*[MAX_N_OBS+1]*/;
__constant__ double CUDA_Phi_0;
__device__ int CUDA_End;

texture<int2,1> texWeight;
texture<int2,1> texbrightness;
texture<int2,1> texsig;

//global to one thread
__device__ freq_context *CUDA_CC;
__device__ freq_result *CUDA_FR;

texture<int2,1> texArea;
texture<int2,1> texDg;

int CUDA_grid_dim;
double *pee,*pee0,*pWeight;

int CUDAPrepare(int cudadev,double *beta_pole,double *lambda_pole,double *par,double cl,double Alamda_start,double Alamda_incr,
	            double ee[][3],double ee0[][3],double *tim,double Phi_0,int checkex,int ndata)
{
	//init gpu
    hipSetDevice(cudadev);
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	//determine gridDim
	hipDeviceProp_t deviceProp;
	int SMXBlock; // Maximum number of resident thread blocks per multiprocessor
    hipGetDeviceProperties(&deviceProp, cudadev);
	if (!checkex)
	{
		auto cudaVersion = CUDA_VERSION;
		fprintf(stderr, "CUDA version: %d\n", cudaVersion);
		//fprintf(stderr, "CUDA RC12!!!!!!!!!!\n");
		fprintf(stderr, "CUDA Device number: %d\n",cudadev);
		fprintf(stderr, "CUDA Device: %s\n",deviceProp.name);
		fprintf(stderr, "Compute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
		//fprintf(stderr, "CUDA Device max grid size(x, y, z): %d, %d, %d \n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		fprintf(stderr, "Multiprocessors: %d\n",deviceProp.multiProcessorCount);
	}

	// NOTE: See this https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities , Table 15.
	// NOTE: Also this https://stackoverflow.com/questions/4391162/cuda-determining-threads-per-block-blocks-per-grid
	// NOTE: NB - Always set MaxUsedRegisters to 32 in order to achieve 100% SM occupancy (project's Configuration properties -> CUDA C/C++ -> Device)
	if (deviceProp.major == 7)
	{
		switch (deviceProp.minor)
		{
			case 0:
			case 2:
				SMXBlock = 32;	// CC 7.0 & 7.2, occupancy 100% = 32 blocks per SMX
			case 5:
				SMXBlock = 16;	// CC 7.5, occupancy 100% = 16 blocks per SMX
			default:
				SMXBlock = 16;	// unknown CC, occupancy unknown, 16 blocks per SMX
		}
	}
	else
	if (deviceProp.major == 6) //CC 6.0, 6.1 & 6.2
	{
		SMXBlock = 32; //occupancy 100% = 32 blocks per SMX
	}
	else
	if (deviceProp.major == 5) //CC 5.0, 5.2 & 5.3
	{
		SMXBlock = 32; //occupancy 100% = 32 blocks per SMX, instead as previous was 16 blocks per SMX which led to only 50%
	}
	else
	if (deviceProp.major == 3) //CC 3.0, 3.2, 3.5 & 3.7
	{
		SMXBlock = 16; //occupancy 100% = 16 blocks per SMX
	}
	/*else
	if (deviceProp.major==2) //CC 2.0 and 2.1
	{
		SMXBlock=8; //occupancy 67% = 8 blocks per SMX
	}
	else
	if ((deviceProp.major==1) && (deviceProp.major==3)) //CC 1.3
	{
		SMXBlock=8; //occupancy 50% = 8 blocks per SMX
		CUDA_BLOCK_DIM=64;
	}*/
	else
	{
		fprintf(stderr, "Unsupported Compute Capability (CC) detected (%d.%d). Supported Compute Capabilities are between 3.0 and 7.5.\n", deviceProp.major, deviceProp.minor);
		return 0;
	}

	CUDA_grid_dim=deviceProp.multiProcessorCount*SMXBlock;

	if (!checkex)
	{
		fprintf(stderr, "Grid dim: %d = %d*%d\n",CUDA_grid_dim,deviceProp.multiProcessorCount,SMXBlock);
		fprintf(stderr, "Block dim: %d\n", CUDA_BLOCK_DIM);
	}

	hipError_t res;

	//Global parameters
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_beta_pole),beta_pole,sizeof(double)*(N_POLES+1));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lambda_pole),lambda_pole,sizeof(double)*(N_POLES+1));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_par),par,sizeof(double)*4);
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cl),&cl,sizeof(cl));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Alamda_start),&Alamda_start,sizeof(Alamda_start));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Alamda_incr),&Alamda_incr,sizeof(Alamda_incr));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Mmax),&m_max,sizeof(m_max));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Lmax),&l_max,sizeof(l_max));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_tim),tim,sizeof(double)*(MAX_N_OBS+1));
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Phi_0),&Phi_0,sizeof(Phi_0));

	res=hipMalloc(&pWeight,(ndata+3+1)*sizeof(double));
	res=hipMemcpy(pWeight,weight,(ndata+3+1)*sizeof(double),hipMemcpyHostToDevice);
	res=hipBindTexture(0, texWeight, pWeight, (ndata+3+1)*sizeof(double));

	res=hipMalloc(&pee,(ndata+1)*3*sizeof(double));
	res=hipMemcpy(pee,ee,(ndata+1)*3*sizeof(double),hipMemcpyHostToDevice);
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ee),&pee,sizeof(void*));

	res=hipMalloc(&pee0,(ndata+1)*3*sizeof(double));
	res=hipMemcpy(pee0,ee0,(ndata+1)*3*sizeof(double),hipMemcpyHostToDevice);
	res=hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ee0),&pee0,sizeof(void*));

	if (res==hipSuccess) return 1; else return 0;
}

void CUDAUnprepare(void)
{
	hipUnbindTexture(texWeight);
	hipFree(pee);
	hipFree(pee0);
	hipFree(pWeight);
}

__global__ void CUDACalculatePrepare(int n_start,int n_max,double freq_start,double freq_step)
{
	int thidx=blockIdx.x;
	int n=n_start+thidx;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
	freq_result *CUDA_LFR=&CUDA_FR[thidx];

	//zero context
//	CUDA_CC is zeroed itself as global memory but need to reset between freq TODO
	if (n>n_max)
	{
        (*CUDA_LCC).isInvalid=1;
		return;
	}
	else
	{
		(*CUDA_LCC).isInvalid=0;
	}

	(*CUDA_LCC).freq = freq_start - (n - 1) * freq_step;

        /* initial poles */
	(*CUDA_LFR).per_best = 0;
	(*CUDA_LFR).dark_best = 0;
	(*CUDA_LFR).la_best = 0;
	(*CUDA_LFR).be_best = 0;
	(*CUDA_LFR).dev_best = 1e40;
}

__global__ void CUDACalculatePreparePole(int m)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
	freq_result *CUDA_LFR=&CUDA_FR[thidx];
	double prd;
	int i;

	if ((*CUDA_LCC).isInvalid)
	{
		atomicAdd(&CUDA_End,1);
		(*CUDA_LFR).isReported=0; //signal not to read result
		return;
	}

	prd = 1 / (*CUDA_LCC).freq;
            /* starts from the initial ellipsoid */
    for (i = 1; i <= CUDA_Ncoef; i++)
       (*CUDA_LCC).cg[i] = CUDA_cg_first[i];

	(*CUDA_LCC).cg[CUDA_Ncoef+1] = CUDA_beta_pole[m];
	(*CUDA_LCC).cg[CUDA_Ncoef+2] = CUDA_lambda_pole[m];

	/* The formulas use beta measured from the pole */
	(*CUDA_LCC).cg[CUDA_Ncoef+1] = 90 - (*CUDA_LCC).cg[CUDA_Ncoef+1];
	/* conversion of lambda, beta to radians */
    (*CUDA_LCC).cg[CUDA_Ncoef+1] = DEG2RAD * (*CUDA_LCC).cg[CUDA_Ncoef+1];
    (*CUDA_LCC).cg[CUDA_Ncoef+2] = DEG2RAD * (*CUDA_LCC).cg[CUDA_Ncoef+2];

    /* Use omega instead of period */
	(*CUDA_LCC).cg[CUDA_Ncoef+3] = 24 * 2 * PI / prd;

    for (i = 1; i <= CUDA_Nphpar; i++)
    {
        (*CUDA_LCC).cg[CUDA_Ncoef+3+i] = CUDA_par[i];
//              ia[Ncoef+3+i] = ia_par[i]; moved to global
   	}
        /* Lommel-Seeliger part */
    (*CUDA_LCC).cg[CUDA_Ncoef+3+CUDA_Nphpar+2] = 1;
        /* Use logarithmic formulation for Lambert to keep it positive */
	(*CUDA_LCC).cg[CUDA_Ncoef+3+CUDA_Nphpar+1] = log(CUDA_cl);

    	/* Levenberg-Marquardt loop */
		// moved to global iter_max,iter_min,iter_dif_max
		//
	(*CUDA_LCC).rchisq = -1;
    (*CUDA_LCC).Alamda = -1;
    (*CUDA_LCC).Niter = 0;
    (*CUDA_LCC).iter_diff = 1e40;
    (*CUDA_LCC).dev_old = 1e30;
    (*CUDA_LCC).dev_new = 0;
//	(*CUDA_LCC).Lastcall=0; always ==0
	(*CUDA_LFR).isReported=0;
}

__global__ void CUDACalculateIter1_Begin(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
	freq_result *CUDA_LFR=&CUDA_FR[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	(*CUDA_LCC).isNiter=(((*CUDA_LCC).Niter < CUDA_n_iter_max) && ((*CUDA_LCC).iter_diff > CUDA_iter_diff_max)) || ((*CUDA_LCC).Niter < CUDA_n_iter_min);

	if ((*CUDA_LCC).isNiter)
    {
		if ((*CUDA_LCC).Alamda<0)
		{
			(*CUDA_LCC).isAlamda=1;
			(*CUDA_LCC).Alamda = CUDA_Alamda_start; /* initial alambda */
		}
		else
			(*CUDA_LCC).isAlamda=0;
	}
	else
	{
        if (!(*CUDA_LFR).isReported)
		{
			atomicAdd(&CUDA_End,1);
			(*CUDA_LFR).isReported=1;
		}
	}

}

__global__ void CUDACalculateIter1_mrqmin1_end(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	/*gauss_err=*/mrqmin_1_end(CUDA_LCC);
}

__global__ void CUDACalculateIter1_mrqmin2_end(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqmin_2_end(CUDA_LCC,CUDA_ia,CUDA_ma);
	(*CUDA_LCC).Niter++;
}

__global__ void CUDACalculateIter1_mrqcof1_start(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_start(CUDA_LCC,(*CUDA_LCC).cg,(*CUDA_LCC).alpha,(*CUDA_LCC).beta);
}

__global__ void CUDACalculateIter1_mrqcof1_matrix(int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_matrix(CUDA_LCC,(*CUDA_LCC).cg,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_curve1(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_curve1(CUDA_LCC,(*CUDA_LCC).cg,(*CUDA_LCC).alpha,(*CUDA_LCC).beta,Inrel,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_curve1_last(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_curve1_last(CUDA_LCC,(*CUDA_LCC).cg,(*CUDA_LCC).alpha,(*CUDA_LCC).beta,Inrel,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_end(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	(*CUDA_LCC).Ochisq=mrqcof_end(CUDA_LCC,(*CUDA_LCC).alpha);
}

__global__ void CUDACalculateIter1_mrqcof2_start(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_start(CUDA_LCC,(*CUDA_LCC).atry,(*CUDA_LCC).covar,(*CUDA_LCC).da);
}

__global__ void CUDACalculateIter1_mrqcof2_matrix(int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_matrix(CUDA_LCC,(*CUDA_LCC).atry,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_curve1(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_curve1(CUDA_LCC,(*CUDA_LCC).atry,(*CUDA_LCC).covar,(*CUDA_LCC).da,Inrel,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_curve1_last(int Inrel,int Lpoints)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_curve1_last(CUDA_LCC,(*CUDA_LCC).atry,(*CUDA_LCC).covar,(*CUDA_LCC).da,Inrel,Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_end(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	(*CUDA_LCC).Chisq=mrqcof_end(CUDA_LCC,(*CUDA_LCC).covar);
}


__global__ void CUDACalculateIter2(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
//	freq_result *CUDA_LFR=&CUDA_FR[thidx];
	int i,j;

	if ((*CUDA_LCC).isInvalid) return;

	if ((*CUDA_LCC).isNiter)
    {
        if (((*CUDA_LCC).Niter == 1) || ((*CUDA_LCC).Chisq < (*CUDA_LCC).Ochisq))
        {
			if (threadIdx.x==0)
			{
				(*CUDA_LCC).Ochisq = (*CUDA_LCC).Chisq;
			}
			__syncthreads();

			int brtmph,brtmpl;
			brtmph=CUDA_Numfac/CUDA_BLOCK_DIM;
			if(CUDA_Numfac%CUDA_BLOCK_DIM) brtmph++;
			brtmpl=threadIdx.x*brtmph;
			brtmph=brtmpl+brtmph;
			if (brtmph>CUDA_Numfac) brtmph=CUDA_Numfac;
			brtmpl++;

			curv(CUDA_LCC,(*CUDA_LCC).cg,brtmpl,brtmph);

			if (threadIdx.x==0)
			{
				for (i = 1; i <= 3; i++)
				{
					(*CUDA_LCC).chck[i] = 0;
					for (j = 1; j <= CUDA_Numfac; j++)
						(*CUDA_LCC).chck[i] = (*CUDA_LCC).chck[i] + (*CUDA_LCC).Area[j] * CUDA_Nor[j][i-1];
				}
				(*CUDA_LCC).rchisq = (*CUDA_LCC).Chisq - (pow((*CUDA_LCC).chck[1],2) + pow((*CUDA_LCC).chck[2],2) + pow((*CUDA_LCC).chck[3],2)) * pow(CUDA_conw_r,2);
			}
        }
		if (threadIdx.x==0)
		{
			(*CUDA_LCC).dev_new = sqrt((*CUDA_LCC).rchisq / (CUDA_ndata - 3));
			/* only if this step is better than the previous,
				1e-10 is for numeric errors */
			if ((*CUDA_LCC).dev_old - (*CUDA_LCC).dev_new > 1e-10)
			{
				(*CUDA_LCC).iter_diff = (*CUDA_LCC).dev_old - (*CUDA_LCC).dev_new;
				(*CUDA_LCC).dev_old = (*CUDA_LCC).dev_new;
			}
	//		(*CUDA_LFR).Niter=(*CUDA_LCC).Niter;
		}
    }
}

__global__ void CUDACalculateFinishPole(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
	freq_result *CUDA_LFR=&CUDA_FR[thidx];
	double totarea,sum,dark,prd,la_tmp,be_tmp;
	int i;

	if ((*CUDA_LCC).isInvalid) return;

    totarea = 0;
    for (i = 1; i <= CUDA_Numfac; i++)
        totarea = totarea + (*CUDA_LCC).Area[i];
    sum = pow((*CUDA_LCC).chck[1],2) + pow((*CUDA_LCC).chck[2],2) + pow((*CUDA_LCC).chck[3],2);
    dark = sqrt(sum);

    /* period solution */
    prd = 2 * PI / (*CUDA_LCC).cg[CUDA_Ncoef+3];

	/* pole solution */
	la_tmp = RAD2DEG * (*CUDA_LCC).cg[CUDA_Ncoef+2];
	be_tmp = 90 - RAD2DEG * (*CUDA_LCC).cg[CUDA_Ncoef+1];

	if ((*CUDA_LCC).dev_new < (*CUDA_LFR).dev_best)
	{
	    (*CUDA_LFR).dev_best = (*CUDA_LCC).dev_new;
	    (*CUDA_LFR).per_best = prd;
	    (*CUDA_LFR).dark_best = dark / totarea * 100;
	    (*CUDA_LFR).la_best = la_tmp;
	    (*CUDA_LFR).be_best = be_tmp;
	}
	//debug
/*	(*CUDA_LFR).dark=dark;
	(*CUDA_LFR).totarea=totarea;
	(*CUDA_LFR).chck[1]=(*CUDA_LCC).chck[1];
	(*CUDA_LFR).chck[2]=(*CUDA_LCC).chck[2];
	(*CUDA_LFR).chck[3]=(*CUDA_LCC).chck[3];*/
}

__global__ void CUDACalculateFinish(void)
{
	int thidx=blockIdx.x;
	freq_context *CUDA_LCC=&CUDA_CC[thidx];
	freq_result *CUDA_LFR=&CUDA_FR[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if ((*CUDA_LFR).la_best < 0)
	   (*CUDA_LFR).la_best += 360;

	if (isnan((*CUDA_LFR).dark_best) == 1)
	    (*CUDA_LFR).dark_best = 1.0;
}

int CUDAPrecalc(double freq_start,double freq_end,double freq_step,double stop_condition,int n_iter_min,double *conw_r,
	            int ndata,int *ia,int *ia_par,int *new_conw,double *cg_first,double *sig,int Numfac,double *brightness)
{
    int max_test_periods,iC,theEnd;
	double sum_dark_facet,ave_dark_facet;
	int i,n,m,n_max=(int) ((freq_start - freq_end) / freq_step) + 1;
	int n_iter_max;
	double iter_diff_max;
	freq_result *res;
	void *pcc,*pfr,*pbrightness,*psig;

     max_test_periods = 10;
     sum_dark_facet = 0.0;
     ave_dark_facet = 0.0;

     if (n_max < max_test_periods)
		max_test_periods = n_max;

    for (i = 1; i <= n_ph_par; i++)
    {
        ia[n_coef+3+i] = ia_par[i];
    }

    n_iter_max = 0;
    iter_diff_max = -1;
    if (stop_condition > 1)
    {
        n_iter_max = (int) stop_condition;
        iter_diff_max = 0;
		n_iter_min = 0; /* to not overwrite the n_iter_max value */
    }
    if (stop_condition < 1)
    {
        n_iter_max = MAX_N_ITER; /* to avoid neverending loop */
        iter_diff_max = stop_condition;
    }

	hipError_t err;

	//here move data to device
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Ncoef),&n_coef,sizeof(n_coef));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nphpar),&n_ph_par,sizeof(n_ph_par));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac),&Numfac,sizeof(Numfac));
	m=Numfac+1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac1),&m,sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ia),ia,sizeof(int)*(MAX_N_PAR+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cg_first),cg_first,sizeof(double)*(MAX_N_PAR+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_max),&n_iter_max,sizeof(n_iter_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_min),&n_iter_min,sizeof(n_iter_min));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ndata),&ndata,sizeof(ndata));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_iter_diff_max),&iter_diff_max,sizeof(iter_diff_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_conw_r),conw_r,sizeof(conw_r));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nor),normal,sizeof(double)*(MAX_N_FAC+1)*3);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fc),f_c,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fs),f_s,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Pleg),pleg,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Darea),d_area,sizeof(double)*(MAX_N_FAC+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dsph),d_sphere,sizeof(double)*(MAX_N_FAC+1)*(MAX_N_PAR+1));

	err=hipMalloc(&pbrightness,(ndata+1)*sizeof(double));
	err=hipMemcpy(pbrightness,brightness,(ndata+1)*sizeof(double),hipMemcpyHostToDevice);
	err=hipBindTexture(0, texbrightness, pbrightness, (ndata+1)*sizeof(double));

	err=hipMalloc(&psig,(ndata+1)*sizeof(double));
	err=hipMemcpy(psig,sig,(ndata+1)*sizeof(double),hipMemcpyHostToDevice);
	err=hipBindTexture(0, texsig, psig, (ndata+1)*sizeof(double));

	/* number of fitted parameters */
	int lmfit=0,llastma=0,llastone=1,ma=n_coef+5+n_ph_par;
		 for (m = 1; m <= ma; m++)
		 {
		  if (ia[m])
		  {
			lmfit++;
			llastma=m;
		  }
		 }
		 llastone=1;
		 for (m = 2; m <=llastma; m++) //ia[1] is skipped because ia[1]=0 is acceptable inside mrqcof
		 {
		  if (!ia[m]) break;
		  llastone=m;
		 }
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ma),&ma,sizeof(ma));
 	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit),&lmfit,sizeof(lmfit));
	m=lmfit+1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit1),&m,sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastma),&llastma,sizeof(llastma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastone),&llastone,sizeof(llastone));
	m=ma-2-n_ph_par;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ncoef0),&m,sizeof(m));

	int CUDA_Grid_dim_precalc=CUDA_grid_dim;
	if (max_test_periods<CUDA_Grid_dim_precalc) CUDA_Grid_dim_precalc=max_test_periods;

	err=hipMalloc(&pcc,CUDA_Grid_dim_precalc*sizeof(freq_context));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_CC),&pcc,sizeof(pcc));
	err=hipMalloc(&pfr,CUDA_Grid_dim_precalc*sizeof(freq_result));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_FR),&pfr,sizeof(pfr));

	m=(Numfac+1)*(n_coef+1);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dg_block),&m,sizeof(m));

    double *pa,*pg,*pal,*pco,*pdytemp,*pytemp;

	err=hipMalloc(&pa,CUDA_Grid_dim_precalc*(Numfac+1)*sizeof(double));
	err=hipBindTexture(0, texArea, pa, CUDA_Grid_dim_precalc*(Numfac+1)*sizeof(double));
	err=hipMalloc(&pg,CUDA_Grid_dim_precalc*(Numfac+1)*(n_coef+1)*sizeof(double));
	err=hipBindTexture(0, texDg, pg, CUDA_Grid_dim_precalc*(Numfac+1)*(n_coef+1)*sizeof(double));
	err=hipMalloc(&pal,CUDA_Grid_dim_precalc*(lmfit+1)*(lmfit+1)*sizeof(double));
	err=hipMalloc(&pco,CUDA_Grid_dim_precalc*(lmfit+1)*(lmfit+1)*sizeof(double));
	err=hipMalloc(&pdytemp,CUDA_Grid_dim_precalc*(max_l_points+1)*(ma+1)*sizeof(double));
	err=hipMalloc(&pytemp,CUDA_Grid_dim_precalc*(max_l_points+1)*sizeof(double));

	for (m=0;m<CUDA_Grid_dim_precalc;m++)
	{
		freq_context ps;
		ps.Area=&pa[m*(Numfac+1)];
		ps.Dg=&pg[m*(Numfac+1)*(n_coef+1)];
		ps.alpha=&pal[m*(lmfit+1)*(lmfit+1)];
		ps.covar=&pco[m*(lmfit+1)*(lmfit+1)];
		ps.dytemp=&pdytemp[m*(max_l_points+1)*(ma+1)];
		ps.ytemp=&pytemp[m*(max_l_points+1)];
		freq_context *pt=&((freq_context *)pcc)[m];
		err=hipMemcpy(pt,&ps,sizeof(void *)*6,hipMemcpyHostToDevice);
	}

	res=(freq_result *)malloc(CUDA_Grid_dim_precalc*sizeof(freq_result));

	for (n=1;n<=max_test_periods;n+=CUDA_Grid_dim_precalc)
	{
        CUDACalculatePrepare<<<CUDA_Grid_dim_precalc,1>>>(n,max_test_periods,freq_start,freq_step);
		err=hipDeviceSynchronize();

		for (m = 1; m <= N_POLES; m++)
		{
			//zero global End signal
			theEnd=0;
			hipMemcpyToSymbol(HIP_SYMBOL(CUDA_End),&theEnd,sizeof(theEnd));
			//
			CUDACalculatePreparePole<<<CUDA_Grid_dim_precalc,1>>>(m);
			//
			while (!theEnd)
			{
				CUDACalculateIter1_Begin<<<CUDA_Grid_dim_precalc,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqcof1_start<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>();
				for (iC=1;iC<l_curves;iC++)
				{
						CUDACalculateIter1_mrqcof1_matrix<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(l_points[iC]);
						CUDACalculateIter1_mrqcof1_curve1<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
						CUDACalculateIter1_mrqcof1_curve2<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
				}
				CUDACalculateIter1_mrqcof1_curve1_last<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof1_curve2<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof1_end<<<CUDA_Grid_dim_precalc,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqmin1_end<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>();
				//mrqcof
				CUDACalculateIter1_mrqcof2_start<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>();
				for (iC=1;iC<l_curves;iC++)
				{
						CUDACalculateIter1_mrqcof2_matrix<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(l_points[iC]);
						CUDACalculateIter1_mrqcof2_curve1<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
						CUDACalculateIter1_mrqcof2_curve2<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
				}
				CUDACalculateIter1_mrqcof2_curve1_last<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof2_curve2<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof2_end<<<CUDA_Grid_dim_precalc,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqmin2_end<<<CUDA_Grid_dim_precalc,1>>>();
				CUDACalculateIter2<<<CUDA_Grid_dim_precalc,CUDA_BLOCK_DIM>>>();
				//err=hipDeviceSynchronize(); memcpy is synchro itself
				hipMemcpyFromSymbol(&theEnd,HIP_SYMBOL(CUDA_End),sizeof(theEnd));
				theEnd=theEnd==CUDA_Grid_dim_precalc;

				//break;//debug
			}
			CUDACalculateFinishPole<<<CUDA_Grid_dim_precalc,1>>>();
			err=hipDeviceSynchronize();
//			err=hipMemcpyFromSymbol(&res,HIP_SYMBOL(CUDA_FR),sizeof(freq_result)*CUDA_Grid_dim_precalc);
//			err=hipMemcpyFromSymbol(&resc,HIP_SYMBOL(CUDA_CC),sizeof(freq_context)*CUDA_Grid_dim_precalc);
			//break; //debug
		}

		CUDACalculateFinish<<<CUDA_Grid_dim_precalc,1>>>();
		//err=hipDeviceSynchronize(); memcpy is synchro itself

		//read results here
		err=hipMemcpy(res,pfr,sizeof(freq_result)*CUDA_Grid_dim_precalc,hipMemcpyDeviceToHost);

		for (m=1; m <= CUDA_Grid_dim_precalc; m++)
		{
		  if (res[m-1].isReported==1)
			sum_dark_facet = sum_dark_facet + res[m-1].dark_best;
		}
   } /* period loop */

	hipUnbindTexture(texArea);
	hipUnbindTexture(texDg);
	hipUnbindTexture(texbrightness);
	hipUnbindTexture(texsig);
	hipFree(pa);
	hipFree(pg);
	hipFree(pal);
	hipFree(pco);
	hipFree(pdytemp);
	hipFree(pytemp);
	hipFree(pcc);
	hipFree(pfr);
	hipFree(pbrightness);
	hipFree(psig);

	free((void *)res);

	ave_dark_facet = sum_dark_facet / max_test_periods;

	if ( ave_dark_facet < 1.0 )
		*new_conw = 1; /* new correct conwexity weight */
	if ( ave_dark_facet >= 1.0 )
		*conw_r = *conw_r * 2; /* still not good */

	return 1;
}

void GetCUDAOccupancy(const int cudaDevice)
{
	int numBlocks;        // Occupancy in terms of active blocks
	const auto blockSize = CUDA_BLOCK_DIM;

	//hipGetDevice(&cudaDevice);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, cudaDevice);

	hipOccupancyMaxActiveBlocksPerMultiprocessor(
		&numBlocks,
		CUDACalculateIter1_mrqcof1_curve2,
		blockSize,
		0);

	const auto activeWarps = numBlocks * blockSize / deviceProp.warpSize;
	const auto maxWarps = deviceProp.maxThreadsPerMultiProcessor / deviceProp.warpSize;

	const auto ocupancy = static_cast<double>(activeWarps) / maxWarps * 100;

	fprintf(stderr, "Occupancy for kernel \"CUDACalculateIter1_mrqcof1_curve2\": %f%%\n", ocupancy);

	//std::cout << "Occupancy: " << (double)activeWarps / maxWarps * 100 << "%" << std::endl;
}

int CUDAStart(int n_start_from,double freq_start,double freq_end,double freq_step,double stop_condition,int n_iter_min,double conw_r,
	          int ndata,int *ia,int *ia_par,double *cg_first,MFILE& mf,double escl,double *sig,int Numfac,double *brightness)
{
	int retval,i,n,m,iC,n_max=(int) ((freq_start - freq_end) / freq_step) + 1;
	int n_iter_max,theEnd,LinesWritten;
	double iter_diff_max;
	freq_result *res;
	void *pcc,*pfr,*pbrightness,*psig;
	char buf[256];

    for (i = 1; i <= n_ph_par; i++)
    {
        ia[n_coef+3+i] = ia_par[i];
    }

    n_iter_max = 0;
    iter_diff_max = -1;
    if (stop_condition > 1)
    {
        n_iter_max = (int) stop_condition;
        iter_diff_max = 0;
		n_iter_min = 0; /* to not overwrite the n_iter_max value */
    }
    if (stop_condition < 1)
    {
        n_iter_max = MAX_N_ITER; /* to avoid neverending loop */
        iter_diff_max = stop_condition;
    }

	hipError_t err;

	//here move data to device
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Ncoef),&n_coef,sizeof(n_coef));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nphpar),&n_ph_par,sizeof(n_ph_par));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac),&Numfac,sizeof(Numfac));
	m=Numfac+1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Numfac1),&m,sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ia),ia,sizeof(int)*(MAX_N_PAR+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_cg_first),cg_first,sizeof(double)*(MAX_N_PAR+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_max),&n_iter_max,sizeof(n_iter_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_n_iter_min),&n_iter_min,sizeof(n_iter_min));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ndata),&ndata,sizeof(ndata));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_iter_diff_max),&iter_diff_max,sizeof(iter_diff_max));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_conw_r),&conw_r,sizeof(conw_r));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Nor),normal,sizeof(double)*(MAX_N_FAC+1)*3);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fc),f_c,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Fs),f_s,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Pleg),pleg,sizeof(double)*(MAX_N_FAC+1)*(MAX_LM+1)*(MAX_LM+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Darea),d_area,sizeof(double)*(MAX_N_FAC+1));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dsph),d_sphere,sizeof(double)*(MAX_N_FAC+1)*(MAX_N_PAR+1));

	err=hipMalloc(&pbrightness,(ndata+1)*sizeof(double));
	err=hipMemcpy(pbrightness,brightness,(ndata+1)*sizeof(double),hipMemcpyHostToDevice);
	err=hipBindTexture(0, texbrightness, pbrightness, (ndata+1)*sizeof(double));

	err=hipMalloc(&psig,(ndata+1)*sizeof(double));
	err=hipMemcpy(psig,sig,(ndata+1)*sizeof(double),hipMemcpyHostToDevice);
	err=hipBindTexture(0, texsig, psig, (ndata+1)*sizeof(double));

	/* number of fitted parameters */
	int lmfit=0,llastma=0,llastone=1,ma=n_coef+5+n_ph_par;
		 for (m = 1; m <= ma; m++)
		 {
		  if (ia[m])
		  {
			lmfit++;
			llastma=m;
		  }
		 }
		 llastone=1;
		 for (m = 2; m <=llastma; m++) //ia[1] is skipped because ia[1]=0 is acceptable inside mrqcof
		 {
		  if (!ia[m]) break;
		  llastone=m;
		 }
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ma),&ma,sizeof(ma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit),&lmfit,sizeof(lmfit));
	m=lmfit+1;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_mfit1),&m,sizeof(m));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastma),&llastma,sizeof(llastma));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_lastone),&llastone,sizeof(llastone));
	m=ma-2-n_ph_par;
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_ncoef0),&m,sizeof(m));

	err=hipMalloc(&pcc,CUDA_grid_dim*sizeof(freq_context));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_CC),&pcc,sizeof(pcc));
	err=hipMalloc(&pfr,CUDA_grid_dim*sizeof(freq_result));
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_FR),&pfr,sizeof(pfr));

	m=(Numfac+1)*(n_coef+1);
	hipMemcpyToSymbol(HIP_SYMBOL(CUDA_Dg_block),&m,sizeof(m));

    double *pa,*pg,*pal,*pco,*pdytemp,*pytemp;

	err=hipMalloc(&pa,CUDA_grid_dim*(Numfac+1)*sizeof(double));
	err=hipBindTexture(0, texArea, pa, CUDA_grid_dim*(Numfac+1)*sizeof(double));
	err=hipMalloc(&pg,CUDA_grid_dim*(Numfac+1)*(n_coef+1)*sizeof(double));
	err=hipBindTexture(0, texDg, pg, CUDA_grid_dim*(Numfac+1)*(n_coef+1)*sizeof(double));
	err=hipMalloc(&pal,CUDA_grid_dim*(lmfit+1)*(lmfit+1)*sizeof(double));
	err=hipMalloc(&pco,CUDA_grid_dim*(lmfit+1)*(lmfit+1)*sizeof(double));
	err=hipMalloc(&pdytemp,CUDA_grid_dim*(max_l_points+1)*(ma+1)*sizeof(double));
	err=hipMalloc(&pytemp,CUDA_grid_dim*(max_l_points+1)*sizeof(double));

	for (m=0;m<CUDA_grid_dim;m++)
	{
		freq_context ps;
		ps.Area=&pa[m*(Numfac+1)];
		ps.Dg=&pg[m*(Numfac+1)*(n_coef+1)];
		ps.alpha=&pal[m*(lmfit+1)*(lmfit+1)];
		ps.covar=&pco[m*(lmfit+1)*(lmfit+1)];
		ps.dytemp=&pdytemp[m*(max_l_points+1)*(ma+1)];
		ps.ytemp=&pytemp[m*(max_l_points+1)];
		freq_context *pt=&((freq_context *)pcc)[m];
		err=hipMemcpy(pt,&ps,sizeof(void *)*6,hipMemcpyHostToDevice);
	}

	res=(freq_result *)malloc(CUDA_grid_dim*sizeof(freq_result));

	int firstreport=0;//beta debug

	for (n=n_start_from;n<=n_max;n+=CUDA_grid_dim)
	{
		auto fractionDone = (double)n / (double)n_max;
		boinc_fraction_done(fractionDone);

//#if _DEBUG
		float fraction = fractionDone * 100;
			printf("Fraction done: %.2f%%\n", fraction);
//#endif

        CUDACalculatePrepare<<<CUDA_grid_dim,1>>>(n,n_max,freq_start,freq_step);
		err=hipDeviceSynchronize();

		for (m = 1; m <= N_POLES; m++)
		{
			//zero global End signal
			theEnd=0;
			hipMemcpyToSymbol(HIP_SYMBOL(CUDA_End),&theEnd,sizeof(theEnd));
			//
			CUDACalculatePreparePole<<<CUDA_grid_dim,1>>>(m);
			//
			while (!theEnd)
			{
				CUDACalculateIter1_Begin<<<CUDA_grid_dim,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqcof1_start<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>();
				for (iC=1;iC<l_curves;iC++)
				{
					CUDACalculateIter1_mrqcof1_matrix<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(l_points[iC]);
					CUDACalculateIter1_mrqcof1_curve1<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
					CUDACalculateIter1_mrqcof1_curve2<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
				}
				CUDACalculateIter1_mrqcof1_curve1_last<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof1_curve2<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof1_end<<<CUDA_grid_dim,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqmin1_end<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>();
				//mrqcof
				CUDACalculateIter1_mrqcof2_start<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>();
				for (iC=1;iC<l_curves;iC++)
				{
					CUDACalculateIter1_mrqcof2_matrix<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(l_points[iC]);
					CUDACalculateIter1_mrqcof2_curve1<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
					CUDACalculateIter1_mrqcof2_curve2<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[iC],l_points[iC]);
				}
				CUDACalculateIter1_mrqcof2_curve1_last<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof2_curve2<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>(in_rel[l_curves],l_points[l_curves]);
				CUDACalculateIter1_mrqcof2_end<<<CUDA_grid_dim,1>>>();
				//mrqcof
				CUDACalculateIter1_mrqmin2_end<<<CUDA_grid_dim,1>>>();
				CUDACalculateIter2<<<CUDA_grid_dim,CUDA_BLOCK_DIM>>>();
				//err=hipDeviceSynchronize(); memcpy is synchro itself
				hipMemcpyFromSymbol(&theEnd,HIP_SYMBOL(CUDA_End),sizeof(theEnd));
				theEnd=theEnd==CUDA_grid_dim;

				//break;//debug
			}
			CUDACalculateFinishPole<<<CUDA_grid_dim,1>>>();
			err=hipDeviceSynchronize();
//			err=hipMemcpyFromSymbol(&res,HIP_SYMBOL(CUDA_FR),sizeof(freq_result)*CUDA_grid_dim);
//			err=hipMemcpyFromSymbol(&resc,HIP_SYMBOL(CUDA_CC),sizeof(freq_context)*CUDA_grid_dim);
			//break; //debug
		}

		CUDACalculateFinish<<<CUDA_grid_dim,1>>>();
		//err=hipDeviceSynchronize(); memcpy is synchro itself

		//read results here
		err=hipMemcpy(res,pfr,sizeof(freq_result)*CUDA_grid_dim,hipMemcpyDeviceToHost);

		LinesWritten=0;
		for (m = 1; m <=CUDA_grid_dim ; m++)
		{
			if (res[m-1].isReported==1)
			{
				LinesWritten++;
				/* output file */
				if (( n ==1 ) && (m==1))
					mf.printf("%.8f  %.6f  %.6f %4.1f %4.0f %4.0f\n", 24 * res[m-1].per_best, res[m-1].dev_best, res[m-1].dev_best * res[m-1].dev_best * (ndata - 3), conw_r * escl * escl, res[m-1].la_best, res[m-1].be_best);
				else
					mf.printf("%.8f  %.6f  %.6f %4.1f %4.0f %4.0f\n", 24 * res[m-1].per_best, res[m-1].dev_best, res[m-1].dev_best * res[m-1].dev_best * (ndata - 3), res[m-1].dark_best, res[m-1].la_best, res[m-1].be_best);
			}
		}
		 if (boinc_time_to_checkpoint() || boinc_is_standalone()) {
			retval = DoCheckpoint(mf, (n-1)+LinesWritten,1,conw_r); //zero lines
			if (retval) {fprintf(stderr, "%s APP: period_search checkpoint failed %d\n", boinc_msg_prefix(buf, sizeof(buf)), retval); exit(retval);}
			boinc_checkpoint_completed();
		 }

//		break;//debug
    } /* period loop */

	hipUnbindTexture(texArea);
	hipUnbindTexture(texDg);
	hipUnbindTexture(texbrightness);
	hipUnbindTexture(texsig);
	hipFree(pa);
	hipFree(pg);
	hipFree(pal);
	hipFree(pco);
	hipFree(pdytemp);
	hipFree(pytemp);
    hipFree(pcc);
	hipFree(pfr);
	hipFree(pbrightness);
	hipFree(psig);

	free((void *)res);

	return 1;
}
