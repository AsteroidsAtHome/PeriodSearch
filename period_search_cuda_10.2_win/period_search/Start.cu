#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

#include "constants.h"
#include "globals_CUDA.h"
#include "declarations_CUDA.h"
#include "../../../../../../../Program Files (x86)/Windows Kits/10/Include/10.0.10240.0/ucrt/math.h"

__global__ void CUDACalculatePrepare(int n_start, int n_max, double freq_start, double freq_step)
{
	int thidx = blockIdx.x;
	int n = n_start + thidx;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	freq_result* CUDA_LFR = &CUDA_FR[thidx];

	//zero context
	//	CUDA_CC is zeroed itself as global memory but need to reset between freq TODO
	if (n > n_max)
	{
		(*CUDA_LCC).isInvalid = 1;
		return;
	}
	else
	{
		(*CUDA_LCC).isInvalid = 0;
	}

	(*CUDA_LCC).freq = freq_start - (n - 1) * freq_step;

	/* initial poles */
	(*CUDA_LFR).per_best = 0;
	(*CUDA_LFR).dark_best = 0;
	(*CUDA_LFR).la_best = 0;
	(*CUDA_LFR).be_best = 0;
	(*CUDA_LFR).dev_best = 1e40;
}

__global__ void CUDACalculatePreparePole(int m)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	freq_result* CUDA_LFR = &CUDA_FR[thidx];
	double prd;
	int i;

	if ((*CUDA_LCC).isInvalid)
	{
		atomicAdd(&CUDA_End, 1);
		(*CUDA_LFR).isReported = 0; //signal not to read result
		return;
	}

	prd = 1 / (*CUDA_LCC).freq;
	/* starts from the initial ellipsoid */
	for (i = 1; i <= CUDA_Ncoef; i++)
		(*CUDA_LCC).cg[i] = CUDA_cg_first[i];

	(*CUDA_LCC).cg[CUDA_Ncoef + 1] = CUDA_beta_pole[m];
	(*CUDA_LCC).cg[CUDA_Ncoef + 2] = CUDA_lambda_pole[m];

	/* The formulas use beta measured from the pole */
	(*CUDA_LCC).cg[CUDA_Ncoef + 1] = 90 - (*CUDA_LCC).cg[CUDA_Ncoef + 1];
	/* conversion of lambda, beta to radians */
	(*CUDA_LCC).cg[CUDA_Ncoef + 1] = DEG2RAD * (*CUDA_LCC).cg[CUDA_Ncoef + 1];
	(*CUDA_LCC).cg[CUDA_Ncoef + 2] = DEG2RAD * (*CUDA_LCC).cg[CUDA_Ncoef + 2];

	/* Use omega instead of period */
	(*CUDA_LCC).cg[CUDA_Ncoef + 3] = 24 * 2 * PI / prd;

	for (i = 1; i <= CUDA_Nphpar; i++)
	{
		(*CUDA_LCC).cg[CUDA_Ncoef + 3 + i] = CUDA_par[i];
		//              ia[Ncoef+3+i] = ia_par[i]; moved to global
	}
	/* Lommel-Seeliger part */
	(*CUDA_LCC).cg[CUDA_Ncoef + 3 + CUDA_Nphpar + 2] = 1;
	/* Use logarithmic formulation for Lambert to keep it positive */
	(*CUDA_LCC).cg[CUDA_Ncoef + 3 + CUDA_Nphpar + 1] = log(CUDA_cl);

	/* Levenberg-Marquardt loop */
	// moved to global iter_max,iter_min,iter_dif_max
	//
	(*CUDA_LCC).rchisq = -1;
	(*CUDA_LCC).Alamda = -1;
	(*CUDA_LCC).Niter = 0;
	(*CUDA_LCC).iter_diff = 1e40;
	(*CUDA_LCC).dev_old = 1e30;
	(*CUDA_LCC).dev_new = 0;
	//	(*CUDA_LCC).Lastcall=0; always ==0
	(*CUDA_LFR).isReported = 0;
}

__global__ void CUDACalculateIter1_Begin(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	freq_result* CUDA_LFR = &CUDA_FR[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	(*CUDA_LCC).isNiter = (((*CUDA_LCC).Niter < CUDA_n_iter_max) && ((*CUDA_LCC).iter_diff > CUDA_iter_diff_max)) || ((*CUDA_LCC).Niter < CUDA_n_iter_min);

	if ((*CUDA_LCC).isNiter)
	{
		if ((*CUDA_LCC).Alamda < 0)
		{
			(*CUDA_LCC).isAlamda = 1;
			(*CUDA_LCC).Alamda = CUDA_Alamda_start; /* initial alambda */
		}
		else
			(*CUDA_LCC).isAlamda = 0;
	}
	else
	{
		if (!(*CUDA_LFR).isReported)
		{
			atomicAdd(&CUDA_End, 1);
			(*CUDA_LFR).isReported = 1;
		}
	}

}

__global__ void CUDACalculateIter1_mrqmin1_end(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	/*gauss_err=*/mrqmin_1_end(CUDA_LCC);
}

__global__ void CUDACalculateIter1_mrqmin2_end(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqmin_2_end(CUDA_LCC, CUDA_ia, CUDA_ma);
	(*CUDA_LCC).Niter++;
}

__global__ void CUDACalculateIter1_mrqcof1_start(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_start(CUDA_LCC, (*CUDA_LCC).cg, (*CUDA_LCC).alpha, (*CUDA_LCC).beta);
}

__global__ void CUDACalculateIter1_mrqcof1_matrix(int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_matrix(CUDA_LCC, (*CUDA_LCC).cg, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_curve1(int Inrel, int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_curve1(CUDA_LCC, (*CUDA_LCC).cg, (*CUDA_LCC).alpha, (*CUDA_LCC).beta, Inrel, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_curve1_last(int Inrel, int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	mrqcof_curve1_last(CUDA_LCC, (*CUDA_LCC).cg, (*CUDA_LCC).alpha, (*CUDA_LCC).beta, Inrel, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof1_end(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	if (!(*CUDA_LCC).isAlamda) return;

	(*CUDA_LCC).Ochisq = mrqcof_end(CUDA_LCC, (*CUDA_LCC).alpha);
}

__global__ void CUDACalculateIter1_mrqcof2_start(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_start(CUDA_LCC, (*CUDA_LCC).atry, (*CUDA_LCC).covar, (*CUDA_LCC).da);
}

__global__ void CUDACalculateIter1_mrqcof2_matrix(int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_matrix(CUDA_LCC, (*CUDA_LCC).atry, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_curve1(int Inrel, int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_curve1(CUDA_LCC, (*CUDA_LCC).atry, (*CUDA_LCC).covar, (*CUDA_LCC).da, Inrel, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_curve1_last(int Inrel, int Lpoints)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	mrqcof_curve1_last(CUDA_LCC, (*CUDA_LCC).atry, (*CUDA_LCC).covar, (*CUDA_LCC).da, Inrel, Lpoints);
}

__global__ void CUDACalculateIter1_mrqcof2_end(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if (!(*CUDA_LCC).isNiter) return;

	(*CUDA_LCC).Chisq = mrqcof_end(CUDA_LCC, (*CUDA_LCC).covar);
}

__global__ void CUDACalculateIter2(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	//	freq_result *CUDA_LFR=&CUDA_FR[thidx];
	int i, j;

	if ((*CUDA_LCC).isInvalid) return;

	if ((*CUDA_LCC).isNiter)
	{
		if (((*CUDA_LCC).Niter == 1) || ((*CUDA_LCC).Chisq < (*CUDA_LCC).Ochisq))
		{
			if (threadIdx.x == 0)
			{
				(*CUDA_LCC).Ochisq = (*CUDA_LCC).Chisq;
			}
			__syncthreads();

			int brtmph, brtmpl;
			brtmph = CUDA_Numfac / CUDA_BLOCK_DIM;
			if (CUDA_Numfac % CUDA_BLOCK_DIM) brtmph++;
			brtmpl = threadIdx.x * brtmph;
			brtmph = brtmpl + brtmph;
			if (brtmph > CUDA_Numfac) brtmph = CUDA_Numfac;
			brtmpl++;

			curv(CUDA_LCC, (*CUDA_LCC).cg, brtmpl, brtmph);

			if (threadIdx.x == 0)
			{
				for (i = 1; i <= 3; i++)
				{
					(*CUDA_LCC).chck[i] = 0;
					for (j = 1; j <= CUDA_Numfac; j++)
						(*CUDA_LCC).chck[i] = (*CUDA_LCC).chck[i] + (*CUDA_LCC).Area[j] * CUDA_Nor[j][i - 1];
				}
				(*CUDA_LCC).rchisq = (*CUDA_LCC).Chisq - (pow((*CUDA_LCC).chck[1], 2) + pow((*CUDA_LCC).chck[2], 2) + pow((*CUDA_LCC).chck[3], 2)) * pow(CUDA_conw_r, 2);
			}
		}
		if (threadIdx.x == 0)
		{
			(*CUDA_LCC).dev_new = sqrt((*CUDA_LCC).rchisq / (CUDA_ndata - 3));
			/* only if this step is better than the previous,
				1e-10 is for numeric errors */
			if ((*CUDA_LCC).dev_old - (*CUDA_LCC).dev_new > 1e-10)
			{
				(*CUDA_LCC).iter_diff = (*CUDA_LCC).dev_old - (*CUDA_LCC).dev_new;
				(*CUDA_LCC).dev_old = (*CUDA_LCC).dev_new;
			}
			//		(*CUDA_LFR).Niter=(*CUDA_LCC).Niter;
		}
	}
}

__global__ void CUDACalculateFinishPole(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	freq_result* CUDA_LFR = &CUDA_FR[thidx];
	double totarea, sum, dark, prd, la_tmp, be_tmp;
	int i;

	if ((*CUDA_LCC).isInvalid) return;

	totarea = 0;
	for (i = 1; i <= CUDA_Numfac; i++)
		totarea = totarea + (*CUDA_LCC).Area[i];
	sum = pow((*CUDA_LCC).chck[1], 2) + pow((*CUDA_LCC).chck[2], 2) + pow((*CUDA_LCC).chck[3], 2);
	dark = sqrt(sum);

	/* period solution */
	prd = 2 * PI / (*CUDA_LCC).cg[CUDA_Ncoef + 3];

	/* pole solution */
	la_tmp = RAD2DEG * (*CUDA_LCC).cg[CUDA_Ncoef + 2];
	be_tmp = 90 - RAD2DEG * (*CUDA_LCC).cg[CUDA_Ncoef + 1];

	if ((*CUDA_LCC).dev_new < (*CUDA_LFR).dev_best)
	{
		(*CUDA_LFR).dev_best = (*CUDA_LCC).dev_new;
		(*CUDA_LFR).per_best = prd;
		(*CUDA_LFR).dark_best = dark / totarea * 100;
		(*CUDA_LFR).la_best = la_tmp;
		(*CUDA_LFR).be_best = be_tmp;
	}
	//debug
	/*	(*CUDA_LFR).dark=dark;
	(*CUDA_LFR).totarea=totarea;
	(*CUDA_LFR).chck[1]=(*CUDA_LCC).chck[1];
	(*CUDA_LFR).chck[2]=(*CUDA_LCC).chck[2];
	(*CUDA_LFR).chck[3]=(*CUDA_LCC).chck[3];*/
}

__global__ void CUDACalculateFinish(void)
{
	int thidx = blockIdx.x;
	freq_context* CUDA_LCC = &CUDA_CC[thidx];
	freq_result* CUDA_LFR = &CUDA_FR[thidx];

	if ((*CUDA_LCC).isInvalid) return;

	if ((*CUDA_LFR).la_best < 0)
		(*CUDA_LFR).la_best += 360;

	if (isnan((*CUDA_LFR).dark_best) == 1)
		(*CUDA_LFR).dark_best = 1.0;
}